#include <stdarg.h>

#include <stdio.h>
#include <stdlib.h>

#include <string.h>
#include <time.h>
#include <math.h>

#include <limits.h>
#include <hip/hip_runtime.h>

#ifndef NULL
#define NULL 0
#endif /* NULL */

#ifndef _WIN32
#include <sys/time.h>
#else /* _WIN32 */
#if !defined(_MSC_EXTENSIONS) && !defined(_INC_WINDOWS)
extern unsigned long __stdcall GetTickCount(void);

#else /* _MSC_EXTENSIONS */
#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#endif /* _MSC_EXTENSIONS */
#endif /* _WIN32 */

#if defined(_MSC_VER) && (_MSC_VER > 1300)
#ifndef FANN_NO_DLL
#define FANN_USE_DLL
#endif /* FANN_USE_LIB */
#endif /* _MSC_VER */
#if defined(_MSC_VER) && (defined(FANN_USE_DLL) || defined(FANN_DLL_EXPORTS))
#ifdef FANN_DLL_EXPORTS
#define FANN_EXTERNAL __declspec(dllexport)
#else /*  */
#define FANN_EXTERNAL __declspec(dllimport)
#endif /* FANN_DLL_EXPORTS*/
#define FANN_API __stdcall
#else /*  */
#define FANN_EXTERNAL
#define FANN_API
#endif /* _MSC_VER */

#define FANN_ERRSTR_MAX 128
typedef double fann_type;


struct fann_error;
struct fann_train_data;

enum fann_errno_enum {
  FANN_E_NO_ERROR = 0,
  FANN_E_CANT_OPEN_CONFIG_R,
  FANN_E_CANT_OPEN_CONFIG_W,
  FANN_E_WRONG_CONFIG_VERSION,
  FANN_E_CANT_READ_CONFIG,
  FANN_E_CANT_READ_NEURON,
  FANN_E_CANT_READ_CONNECTIONS,
  FANN_E_WRONG_NUM_CONNECTIONS,
  FANN_E_CANT_OPEN_TD_W,
  FANN_E_CANT_OPEN_TD_R,
  FANN_E_CANT_READ_TD,
  FANN_E_CANT_ALLOCATE_MEM,
  FANN_E_CANT_TRAIN_ACTIVATION,
  FANN_E_CANT_USE_ACTIVATION,
  FANN_E_TRAIN_DATA_MISMATCH,
  FANN_E_CANT_USE_TRAIN_ALG,
  FANN_E_TRAIN_DATA_SUBSET,
  FANN_E_INDEX_OUT_OF_BOUND,
  FANN_E_SCALE_NOT_PRESENT,
  FANN_E_INPUT_NO_MATCH,
  FANN_E_OUTPUT_NO_MATCH,
  FANN_E_WRONG_PARAMETERS_FOR_CREATE
};

struct fann_error {
  enum fann_errno_enum errno_f;
  FILE *error_log;
  char *errstr;
};

/* called fann_max, in order to not interferre with predefined versions of max */
#define fann_max(x, y) (((x) > (y)) ? (x) : (y))
#define fann_min(x, y) (((x) < (y)) ? (x) : (y))
#define fann_safe_free(x) \
  {                       \
    if (x) {              \
      free(x);            \
      x = NULL;           \
    }                     \
  }
#define fann_clip(x, lo, hi) (((x) < (lo)) ? (lo) : (((x) > (hi)) ? (hi) : (x)))
#define fann_exp2(x) exp(0.69314718055994530942 * (x))
/*#define fann_clip(x, lo, hi) (x)*/

#define fann_rand(min_value, max_value) \
  (((float)(min_value)) +               \
   (((float)(max_value) - ((float)(min_value))) * rand() / (RAND_MAX + 1.0f)))

#define fann_abs(value) (((value) > 0) ? (value) : -(value))


#define fann_mult(x, y) (x * y)
#define fann_div(x, y) (x / y)
#define fann_random_weight() (fann_rand(-0.1f, 0.1f))
#define fann_random_bias_weight() (fann_rand(-0.1f, 0.1f))



enum fann_train_enum {
  FANN_TRAIN_INCREMENTAL = 0,
  FANN_TRAIN_BATCH,
  FANN_TRAIN_RPROP,
  FANN_TRAIN_QUICKPROP,
  FANN_TRAIN_SARPROP
};

/* Constant: FANN_TRAIN_NAMES
   Constant array consisting of the names for the training algorithms, so that the name of an
   training function can be received by:
   (code)
   char *name = FANN_TRAIN_NAMES[train_function];
   (end)
   See Also:
      <fann_train_enum>
*/
static char const *const FANN_TRAIN_NAMES[] = {"FANN_TRAIN_INCREMENTAL", "FANN_TRAIN_BATCH",
                                               "FANN_TRAIN_RPROP", "FANN_TRAIN_QUICKPROP",
                                               "FANN_TRAIN_SARPROP"};

   
/* Group: Error Handling */

/* Function: fann_set_error_log
   Change where errors are logged to. Both <struct fann> and <struct fann_data> can be
   casted to <struct fann_error>, so this function can be used to set either of these.
   If log_file is NULL, no errors will be printed.
   If errdat is NULL, the default log will be set. The default log is the log used when creating
   <struct fann> and <struct fann_data>. This default log will also be the default for all new
   structs that are created.
   The default behavior is to log them to stderr.
   See also:
    <struct fann_error>
   This function appears in FANN >= 1.1.0.
 */
  void FANN_API fann_set_error_log(struct fann_error *errdat, FILE *log_file);

/* Function: fann_get_errno
   Returns the last error number.
   See also:
    <fann_errno_enum>, <fann_reset_errno>
   This function appears in FANN >= 1.1.0.
 */
  enum fann_errno_enum FANN_API fann_get_errno(struct fann_error *errdat);

/* Function: fann_reset_errno
   Resets the last error number.
   This function appears in FANN >= 1.1.0.
 */
  void FANN_API fann_reset_errno(struct fann_error *errdat);

/* Function: fann_reset_errstr
   Resets the last error string.
   This function appears in FANN >= 1.1.0.
 */
  void FANN_API fann_reset_errstr(struct fann_error *errdat);

/* Function: fann_get_errstr
   Returns the last errstr.
   This function calls <fann_reset_errno> and <fann_reset_errstr>
   This function appears in FANN >= 1.1.0.
 */
  char *FANN_API fann_get_errstr(struct fann_error *errdat);

/* Function: fann_print_error
   Prints the last error to stderr.
   This function appears in FANN >= 1.1.0.
 */
  void FANN_API fann_print_error(struct fann_error *errdat);

  extern FILE *fann_default_error_log;


/* resets the last error number
 */
    void FANN_API fann_reset_errno(struct fann_error *errdat) {
  errdat->errno_f = FANN_E_NO_ERROR;
}

/* resets the last errstr
 */
    void FANN_API fann_reset_errstr(struct fann_error *errdat) {
  if (errdat->errstr != NULL) free(errdat->errstr);
  errdat->errstr = NULL;
}

/* returns the last error number
 */
    enum fann_errno_enum FANN_API fann_get_errno(struct fann_error *errdat) {
  return errdat->errno_f;
}

/* returns the last errstr
 */
    char *FANN_API fann_get_errstr(struct fann_error *errdat) {
  char *errstr = errdat->errstr;

  fann_reset_errno(errdat);
  fann_reset_errstr(errdat);

  return errstr;
}


/* prints the last error to stderr
 */
    void FANN_API fann_print_error(struct fann_error *errdat) {
  if (errdat->errno_f != FANN_E_NO_ERROR && errdat->errstr != NULL) {
    fprintf(stderr, "FANN Error %d: %s", errdat->errno_f, errdat->errstr);
  }
}


#if defined(_WIN32) && !defined(__MINGW32__)
#define PATH_MAX _MAX_PATH
#endif
#ifndef PATH_MAX
#ifdef _POSIX_PATH_MAX
#define PATH_MAX _POSIX_PATH_MAX
#else
#define PATH_MAX 4096
#endif
#endif

/* INTERNAL FUNCTION
   Populate the error information
 */

FILE *FANN_API fann_default_error_log = (FILE *)-1;

void fann_error(struct fann_error *errdat, const enum fann_errno_enum errno_f, ...) {
  va_list ap;
  size_t errstr_max = FANN_ERRSTR_MAX + PATH_MAX - 1;
  char errstr[FANN_ERRSTR_MAX + PATH_MAX];
  FILE *error_log = fann_default_error_log;

  if (errdat != NULL) errdat->errno_f = errno_f;

  va_start(ap, errno_f);
  switch (errno_f) {
    case FANN_E_NO_ERROR:
      return;
    case FANN_E_CANT_OPEN_CONFIG_R:
      vsnprintf(errstr, errstr_max, "Unable to open configuration file \"%s\" for reading.\n", ap);
      break;
    case FANN_E_CANT_OPEN_CONFIG_W:
      vsnprintf(errstr, errstr_max, "Unable to open configuration file \"%s\" for writing.\n", ap);
      break;
    case FANN_E_WRONG_CONFIG_VERSION:
      vsnprintf(
          errstr, errstr_max,
          "Wrong version of configuration file, aborting read of configuration file \"%s\".\n", ap);
      break;
    case FANN_E_CANT_READ_CONFIG:
      vsnprintf(errstr, errstr_max, "Error reading \"%s\" from configuration file \"%s\".\n", ap);
      break;
    case FANN_E_CANT_READ_NEURON:
      vsnprintf(errstr, errstr_max, "Error reading neuron info from configuration file \"%s\".\n",
                ap);
      break;
    case FANN_E_CANT_READ_CONNECTIONS:
      vsnprintf(errstr, errstr_max, "Error reading connections from configuration file \"%s\".\n",
                ap);
      break;
    case FANN_E_WRONG_NUM_CONNECTIONS:
      vsnprintf(errstr, errstr_max, "ERROR connections_so_far=%d, total_connections=%d\n", ap);
      break;
    case FANN_E_CANT_OPEN_TD_W:
      vsnprintf(errstr, errstr_max, "Unable to open train data file \"%s\" for writing.\n", ap);
      break;
    case FANN_E_CANT_OPEN_TD_R:
      vsnprintf(errstr, errstr_max, "Unable to open train data file \"%s\" for writing.\n", ap);
      break;
    case FANN_E_CANT_READ_TD:
      vsnprintf(errstr, errstr_max, "Error reading info from train data file \"%s\", line: %d.\n",
                ap);
      break;
    case FANN_E_CANT_ALLOCATE_MEM:
      strcpy(errstr, "Unable to allocate memory.\n");
      break;
    case FANN_E_CANT_TRAIN_ACTIVATION:
      strcpy(errstr, "Unable to train with the selected activation function.\n");
      break;
    case FANN_E_CANT_USE_ACTIVATION:
      strcpy(errstr, "Unable to use the selected activation function.\n");
      break;
    case FANN_E_TRAIN_DATA_MISMATCH:
      strcpy(errstr, "Training data must be of equivalent structure.\n");
      break;
    case FANN_E_CANT_USE_TRAIN_ALG:
      strcpy(errstr, "Unable to use the selected training algorithm.\n");
      break;
    case FANN_E_TRAIN_DATA_SUBSET:
      vsnprintf(errstr, errstr_max,
                "Subset from %d of length %d not valid in training set of length %d.\n", ap);
      break;
    case FANN_E_INDEX_OUT_OF_BOUND:
      vsnprintf(errstr, errstr_max, "Index %d is out of bound.\n", ap);
      break;
    case FANN_E_SCALE_NOT_PRESENT:
      strcpy(errstr, "Scaling parameters not present.\n");
      break;
    case FANN_E_INPUT_NO_MATCH:
      vsnprintf(errstr, errstr_max,
                "The number of input neurons in the ann (%d) and data (%d) don't match\n", ap);
      break;
    case FANN_E_OUTPUT_NO_MATCH:
      vsnprintf(errstr, errstr_max,
                "The number of output neurons in the ann (%d) and data (%d) don't match\n", ap);
      break;
    case FANN_E_WRONG_PARAMETERS_FOR_CREATE:
      strcpy(errstr,
             "The parameters for create_standard are wrong, either too few parameters provided or "
             "a negative/very high value provided.\n");
      break;
  }
  va_end(ap);

  if (errdat != NULL) {
    if (errdat->errstr == NULL) {
      errdat->errstr = (char *)malloc(strlen(errstr) + 1);
    } else if (strlen(errdat->errstr) < strlen(errstr)) {
      errdat->errstr = (char *)realloc(errdat->errstr, strlen(errstr) + 1);
    }
    /* allocation failed */
    if (errdat->errstr == NULL) {
      fprintf(stderr, "Unable to allocate memory.\n");
      return;
    }
    strcpy(errdat->errstr, errstr);
    error_log = errdat->error_log;
  }

  if (error_log == (FILE *)-1) /* This is the default behavior and will give stderr */
  {
    fprintf(stderr, "FANN Error %d: %s", errno_f, errstr);
  } else if (error_log != NULL) {
    fprintf(error_log, "FANN Error %d: %s", errno_f, errstr);
  }
}


      
/* INTERNAL FUNCTION
   Initialize an error data strcuture
 */
void fann_init_error_data(struct fann_error *errdat) {
  errdat->errstr = NULL;
  errdat->errno_f = FANN_E_NO_ERROR;
  errdat->error_log = fann_default_error_log;
}

enum fann_activationfunc_enum {
  FANN_LINEAR = 0,
  FANN_THRESHOLD,
  FANN_THRESHOLD_SYMMETRIC,
  FANN_SIGMOID,
  FANN_SIGMOID_STEPWISE,
  FANN_SIGMOID_SYMMETRIC,
  FANN_SIGMOID_SYMMETRIC_STEPWISE,
  FANN_GAUSSIAN,
  FANN_GAUSSIAN_SYMMETRIC,
  /* Stepwise linear approximation to gaussian.
   * Faster than gaussian but a bit less precise.
   * NOT implemented yet.
   */
  FANN_GAUSSIAN_STEPWISE,
  FANN_ELLIOT,
  FANN_ELLIOT_SYMMETRIC,
  FANN_LINEAR_PIECE,
  FANN_LINEAR_PIECE_SYMMETRIC,
  FANN_SIN_SYMMETRIC,
  FANN_COS_SYMMETRIC,
  FANN_SIN,
  FANN_COS
};

/* Constant: FANN_ACTIVATIONFUNC_NAMES
   Constant array consisting of the names for the activation function, so that the name of an
   activation function can be received by:
   (code)
   char *name = FANN_ACTIVATIONFUNC_NAMES[activation_function];
   (end)
   See Also:
      <fann_activationfunc_enum>
*/

#define FANN_EXP(x) exp(x)
#define FANN_SIN(x) sin(x)
#define FANN_COS(x) cos(x)

#define fann_linear_func(v1, r1, v2, r2, sum) \
  (((((r2) - (r1)) * ((sum) - (v1))) / ((v2) - (v1))) + (r1))
#define fann_stepwise(v1, v2, v3, v4, v5, v6, r1, r2, r3, r4, r5, r6, min, max, sum)           \
  (sum < v5 ? (sum < v3 ? (sum < v2 ? (sum < v1 ? min : fann_linear_func(v1, r1, v2, r2, sum)) \
                                    : fann_linear_func(v2, r2, v3, r3, sum))                   \
                        : (sum < v4 ? fann_linear_func(v3, r3, v4, r4, sum)                    \
                                    : fann_linear_func(v4, r4, v5, r5, sum)))                  \
            : (sum < v6 ? fann_linear_func(v5, r5, v6, r6, sum) : max))

/* FANN_LINEAR */
/* #define fann_linear(steepness, sum) fann_mult(steepness, sum) */
#define fann_linear_derive(steepness, value) (steepness)

/* FANN_SIGMOID */
/* #define fann_sigmoid(steepness, sum) (1.0f/(1.0f + exp(-2.0f * steepness * sum))) */
#define fann_sigmoid_real(sum) (1.0f / (1.0f + FANN_EXP(-2.0f * sum)))
#define fann_sigmoid_derive(steepness, value) (2.0f * steepness * value * (1.0f - value))

/* FANN_SIGMOID_SYMMETRIC */
/* #define fann_sigmoid_symmetric(steepness, sum) (2.0f/(1.0f + exp(-2.0f * steepness * sum))
 * - 1.0f) */
#define fann_sigmoid_symmetric_real(sum) (2.0f / (1.0f + FANN_EXP(-2.0f * sum)) - 1.0f)
#define fann_sigmoid_symmetric_derive(steepness, value) steepness*(1.0f - (value * value))

/* FANN_GAUSSIAN */
/* #define fann_gaussian(steepness, sum) (exp(-sum * steepness * sum * steepness)) */
#define fann_gaussian_real(sum) (FANN_EXP(-sum * sum))
#define fann_gaussian_derive(steepness, value, sum) (-2.0f * sum * value * steepness * steepness)

/* FANN_GAUSSIAN_SYMMETRIC */
/* #define fann_gaussian_symmetric(steepness, sum) ((exp(-sum * steepness * sum *
 * steepness)*2.0)-1.0) */
#define fann_gaussian_symmetric_real(sum) ((FANN_EXP(-sum * sum) * 2.0f) - 1.0f)
#define fann_gaussian_symmetric_derive(steepness, value, sum) \
  (-2.0f * sum * (value + 1.0f) * steepness * steepness)

/* FANN_ELLIOT */
/* #define fann_elliot(steepness, sum) (((sum * steepness) / 2.0f) / (1.0f + fann_abs(sum *
 * steepness)) + 0.5f) */
#define fann_elliot_real(sum) (((sum) / 2.0f) / (1.0f + fann_abs(sum)) + 0.5f)
#define fann_elliot_derive(steepness, value, sum) \
  (steepness * 1.0f / (2.0f * (1.0f + fann_abs(sum)) * (1.0f + fann_abs(sum))))

/* FANN_ELLIOT_SYMMETRIC */
/* #define fann_elliot_symmetric(steepness, sum) ((sum * steepness) / (1.0f + fann_abs(sum *
 * steepness)))*/
#define fann_elliot_symmetric_real(sum) ((sum) / (1.0f + fann_abs(sum)))
#define fann_elliot_symmetric_derive(steepness, value, sum) \
  (steepness * 1.0f / ((1.0f + fann_abs(sum)) * (1.0f + fann_abs(sum))))

/* FANN_SIN_SYMMETRIC */
#define fann_sin_symmetric_real(sum) (FANN_SIN(sum))
#define fann_sin_symmetric_derive(steepness, sum) (steepness * cos(steepness * sum))

/* FANN_COS_SYMMETRIC */
#define fann_cos_symmetric_real(sum) (FANN_COS(sum))
#define fann_cos_symmetric_derive(steepness, sum) (steepness * -sin(steepness * sum))

/* FANN_SIN */
#define fann_sin_real(sum) (FANN_SIN(sum) / 2.0f + 0.5f)
#define fann_sin_derive(steepness, sum) (steepness * cos(steepness * sum) / 2.0f)

/* FANN_COS */
#define fann_cos_real(sum) (FANN_COS(sum) / 2.0f + 0.5f)
#define fann_cos_derive(steepness, sum) (steepness * -sin(steepness * sum) / 2.0f)

#define fann_activation_switch(activation_function, value, result)                                 \
  switch (activation_function) {                                                                   \
    case FANN_LINEAR:                                                                              \
      result = (fann_type)value;                                                                   \
      break;                                                                                       \
    case FANN_LINEAR_PIECE:                                                                        \
      result = (fann_type)((value < 0) ? 0 : (value > 1) ? 1 : value);                             \
      break;                                                                                       \
    case FANN_LINEAR_PIECE_SYMMETRIC:                                                              \
      result = (fann_type)((value < -1) ? -1 : (value > 1) ? 1 : value);                           \
      break;                                                                                       \
    case FANN_SIGMOID:                                                                             \
      result = (fann_type)fann_sigmoid_real(value);                                                \
      break;                                                                                       \
    case FANN_SIGMOID_SYMMETRIC:                                                                   \
      result = (fann_type)fann_sigmoid_symmetric_real(value);                                      \
      break;                                                                                       \
    case FANN_SIGMOID_SYMMETRIC_STEPWISE:                                                          \
      result = (fann_type)fann_stepwise(                                                           \
          ((fann_type)-2.64665293693542480469e+00), ((fann_type)-1.47221934795379638672e+00),      \
          ((fann_type)-5.49306154251098632812e-01), ((fann_type)5.49306154251098632812e-01),       \
          ((fann_type)1.47221934795379638672e+00), ((fann_type)2.64665293693542480469e+00),        \
          ((fann_type)-9.90000009536743164062e-01), ((fann_type)-8.99999976158142089844e-01),      \
          ((fann_type)-5.00000000000000000000e-01), ((fann_type)5.00000000000000000000e-01),       \
          ((fann_type)8.99999976158142089844e-01), ((fann_type)9.90000009536743164062e-01), -1, 1, \
          value);                                                                                  \
      break;                                                                                       \
    case FANN_SIGMOID_STEPWISE:                                                                    \
      result = (fann_type)fann_stepwise(                                                           \
          ((fann_type)-2.64665246009826660156e+00), ((fann_type)-1.47221946716308593750e+00),      \
          ((fann_type)-5.49306154251098632812e-01), ((fann_type)5.49306154251098632812e-01),       \
          ((fann_type)1.47221934795379638672e+00), ((fann_type)2.64665293693542480469e+00),        \
          ((fann_type)4.99999988824129104614e-03), ((fann_type)5.00000007450580596924e-02),        \
          ((fann_type)2.50000000000000000000e-01), ((fann_type)7.50000000000000000000e-01),        \
          ((fann_type)9.49999988079071044922e-01), ((fann_type)9.95000004768371582031e-01), 0, 1,  \
          value);                                                                                  \
      break;                                                                                       \
    case FANN_THRESHOLD:                                                                           \
      result = (fann_type)((value < 0) ? 0 : 1);                                                   \
      break;                                                                                       \
    case FANN_THRESHOLD_SYMMETRIC:                                                                 \
      result = (fann_type)((value < 0) ? -1 : 1);                                                  \
      break;                                                                                       \
    case FANN_GAUSSIAN:                                                                            \
      result = (fann_type)fann_gaussian_real(value);                                               \
      break;                                                                                       \
    case FANN_GAUSSIAN_SYMMETRIC:                                                                  \
      result = (fann_type)fann_gaussian_symmetric_real(value);                                     \
      break;                                                                                       \
    case FANN_ELLIOT:                                                                              \
      result = (fann_type)fann_elliot_real(value);                                                 \
      break;                                                                                       \
    case FANN_ELLIOT_SYMMETRIC:                                                                    \
      result = (fann_type)fann_elliot_symmetric_real(value);                                       \
      break;                                                                                       \
    case FANN_SIN_SYMMETRIC:                                                                       \
      result = (fann_type)fann_sin_symmetric_real(value);                                          \
      break;                                                                                       \
    case FANN_COS_SYMMETRIC:                                                                       \
      result = (fann_type)fann_cos_symmetric_real(value);                                          \
      break;                                                                                       \
    case FANN_SIN:                                                                                 \
      result = (fann_type)fann_sin_real(value);                                                    \
      break;                                                                                       \
    case FANN_COS:                                                                                 \
      result = (fann_type)fann_cos_real(value);                                                    \
      break;                                                                                       \
    case FANN_GAUSSIAN_STEPWISE:                                                                   \
      result = 0;                                                                                  \
      break;                                                                                       \
  }

/* Enum: fann_errorfunc_enum
        Error function used during training.
        FANN_ERRORFUNC_LINEAR - Standard linear error function.
        FANN_ERRORFUNC_TANH - Tanh error function, usually better
                but can require a lower learning rate. This error function aggressively targets
   outputs that differ much from the desired, while not targeting outputs that only differ a little
   that much. This activation function is not recommended for cascade training and incremental
   training.
        See also:
                <fann_set_train_error_function>, <fann_get_train_error_function>
*/


enum fann_errorfunc_enum { FANN_ERRORFUNC_LINEAR = 0, FANN_ERRORFUNC_TANH };

/* Constant: FANN_ERRORFUNC_NAMES
   Constant array consisting of the names for the training error functions, so that the name of an
   error function can be received by:
   (code)
   char *name = FANN_ERRORFUNC_NAMES[error_function];
   (end)
   See Also:
      <fann_errorfunc_enum>
*/
static char const *const FANN_ERRORFUNC_NAMES[] = {"FANN_ERRORFUNC_LINEAR", "FANN_ERRORFUNC_TANH"};

/* Enum: fann_stopfunc_enum
        Stop criteria used during training.
        FANN_STOPFUNC_MSE - Stop criterion is Mean Square Error (MSE) value.
        FANN_STOPFUNC_BIT - Stop criterion is number of bits that fail. The number of bits; means
   the number of output neurons which differ more than the bit fail limit (see
   <fann_get_bit_fail_limit>, <fann_set_bit_fail_limit>). The bits are counted in all of the
   training data, so this number can be higher than the number of training data.
        See also:
                <fann_set_train_stop_function>, <fann_get_train_stop_function>
*/
enum fann_stopfunc_enum { FANN_STOPFUNC_MSE = 0, FANN_STOPFUNC_BIT };

/* Constant: FANN_STOPFUNC_NAMES
   Constant array consisting of the names for the training stop functions, so that the name of a
   stop function can be received by:
   (code)
   char *name = FANN_STOPFUNC_NAMES[stop_function];
   (end)
   See Also:
      <fann_stopfunc_enum>
*/
static char const *const FANN_STOPFUNC_NAMES[] = {"FANN_STOPFUNC_MSE", "FANN_STOPFUNC_BIT"};

/* Enum: fann_network_type_enum
    Definition of network types used by <fann_get_network_type>
    FANN_NETTYPE_LAYER - Each layer only has connections to the next layer
    FANN_NETTYPE_SHORTCUT - Each layer has connections to all following layers
   See Also:
      <fann_get_network_type>
   This enumeration appears in FANN >= 2.1.0
*/
enum fann_nettype_enum {
  FANN_NETTYPE_LAYER = 0, /* Each layer only has connections to the next layer */
  FANN_NETTYPE_SHORTCUT   /* Each layer has connections to all following layers */
};

/* Constant: FANN_NETWORK_TYPE_NAMES
   Constant array consisting of the names for the network types, so that the name of an
   network type can be received by:
   (code)
   char *network_type_name = FANN_NETWORK_TYPE_NAMES[fann_get_network_type(ann)];
   (end)
   See Also:
      <fann_get_network_type>
   This constant appears in FANN >= 2.1.0
*/
static char const *const FANN_NETTYPE_NAMES[] = {"FANN_NETTYPE_LAYER", "FANN_NETTYPE_SHORTCUT"};

/* forward declarations for use with the callback */
struct fann;


struct fann_train_data {
  enum fann_errno_enum errno_f;
  FILE *error_log;
  char *errstr;

  unsigned int num_data;
  unsigned int num_input;
  unsigned int num_output;
  fann_type **input;
  fann_type **output;
};
/* Type: fann_callback_type
   This callback function can be called during training when using <fann_train_on_data>,
   <fann_train_on_file> or <fann_cascadetrain_on_data>.
        >typedef int (FANN_API * fann_callback_type) (struct fann *ann, struct fann_train_data
   *train,
        > unsigned int max_epochs, >                                             unsigned int
   epochs_between_reports, >                                             float desired_error,
   unsigned int epochs);
        The callback can be set by using <fann_set_callback> and is very useful for doing custom
        things during training. It is recommended to use this function when implementing custom
        training procedures, or when visualizing the training in a GUI etc. The parameters which the
        callback function takes are the parameters given to <fann_train_on_data>, plus an epochs
        parameter which tells how many epochs the training has taken so far.
        The callback function should return an integer, if the callback function returns -1, the
   training will terminate.
        Example of a callback function:
                >int FANN_API test_callback(struct fann *ann, struct fann_train_data *train,
                >				            unsigned int max_epochs, unsigned int
   epochs_between_reports,
                >				            float desired_error, unsigned int
   epochs)
                >{
                >	printf("Epochs     %8d. MSE: %.5f. Desired-MSE: %.5f\n", epochs,
   fann_get_MSE(ann), desired_error); >	return 0;
                >}
        See also:
                <fann_set_callback>, <fann_train_on_data>
 */
  typedef int(FANN_API *fann_callback_type)(struct fann *ann,
                                                        struct fann_train_data *train,
                                                        unsigned int max_epochs,
                                                        unsigned int epochs_between_reports,
                                                        float desired_error, unsigned int epochs);

/* ----- Data structures -----
 * No data within these structures should be altered directly by the user.
 */

struct fann_neuron {
  /* Index to the first and last connection
   * (actually the last is a past end index)
   */
  unsigned int first_con;
  unsigned int last_con;
  /* The sum of the inputs multiplied with the weights */
  fann_type sum;
  /* The value of the activation function applied to the sum */
  fann_type value;
  /* The steepness of the activation function */
  fann_type activation_steepness;
  /* Used to choose which activation function to use */
  enum fann_activationfunc_enum activation_function;

};


/* A single layer in the neural network.
 */
struct fann_layer {
  /* A pointer to the first neuron in the layer
   * When allocated, all the neurons in all the layers are actually
   * in one long array, this is because we want to easily clear all
   * the neurons at once.
   */
  struct fann_neuron *first_neuron;

  /* A pointer to the neuron past the last neuron in the layer */
  /* the number of neurons is last_neuron - first_neuron */
  struct fann_neuron *last_neuron;
};

/* Struct: struct fann_error
        Structure used to store error-related information, both
        <struct fann> and <struct fann_train_data> can be casted to this type.
        See also:
                <fann_set_error_log>, <fann_get_errno>
*/


/* 	Struct: struct fann
        The fast artificial neural network (fann) structure.
        Data within this structure should never be accessed directly, but only by using the
        *fann_get_...* and *fann_set_...* functions.
        The fann structure is created using one of the *fann_create_...* functions and each of
        the functions which operates on the structure takes *struct fann * ann* as the first
   parameter.
        See also:
                <fann_create_standard>, <fann_destroy>
 */
struct fann {
  /* The type of error that last occured. */
  enum fann_errno_enum errno_f;

  /* Where to log error messages. */
  FILE *error_log;

  /* A string representation of the last error. */
  char *errstr;

  /* the learning rate of the network */
  float learning_rate;

  /* The learning momentum used for backpropagation algorithm. */
  float learning_momentum;

  /* the connection rate of the network
   * between 0 and 1, 1 meaning fully connected
   */
  float connection_rate;

  /* is 1 if shortcut connections are used in the ann otherwise 0
   * Shortcut connections are connections that skip layers.
   * A fully connected ann with shortcut connections are a ann where
   * neurons have connections to all neurons in all later layers.
   */
  enum fann_nettype_enum network_type;

  /* pointer to the first layer (input layer) in an array af all the layers,
   * including the input and outputlayers
   */
  struct fann_layer *first_layer;

  /* pointer to the layer past the last layer in an array af all the layers,
   * including the input and outputlayers
   */
  struct fann_layer *last_layer;

  /* Total number of neurons.
   * very useful, because the actual neurons are allocated in one long array
   */
  unsigned int total_neurons;

  /* Number of input neurons (not calculating bias) */
  unsigned int num_input;

  /* Number of output neurons (not calculating bias) */
  unsigned int num_output;

  /* The weight array */
  fann_type *weights;

  /* The connection array */
  struct fann_neuron **connections;

  /* Used to contain the errors used during training
   * Is allocated during first training session,
   * which means that if we do not train, it is never allocated.
   */
  fann_type *train_errors;

  /* Training algorithm used when calling fann_train_on_..
   */
  enum fann_train_enum training_algorithm;

  /* Total number of connections.
   * very useful, because the actual connections
   * are allocated in one long array
   */
  unsigned int total_connections;

  /* used to store outputs in */
  fann_type *output;

  /* the number of data used to calculate the mean square error.
   */
  unsigned int num_MSE;

  /* the total error value.
   * the real mean square error is MSE_value/num_MSE
   */
  float MSE_value;

  /* The number of outputs which would fail (only valid for classification problems)
   */
  unsigned int num_bit_fail;

  /* The maximum difference between the actual output and the expected output
   * which is accepted when counting the bit fails.
   * This difference is multiplied by two when dealing with symmetric activation functions,
   * so that symmetric and not symmetric activation functions can use the same limit.
   */
  fann_type bit_fail_limit;

  /* The error function used during training. (default FANN_ERRORFUNC_TANH)
   */
  enum fann_errorfunc_enum train_error_function;

  /* The stop function used during training. (default FANN_STOPFUNC_MSE)
   */
  enum fann_stopfunc_enum train_stop_function;

  /* The callback function used during training. (default NULL)
   */
  fann_callback_type callback;

  /* A pointer to user defined data. (default NULL)
   */
  void *user_data;

  /* Variables for use with Cascade Correlation */

  /* The error must change by at least this
   * fraction of its old value to count as a
   * significant change.
   */
  float cascade_output_change_fraction;

  /* No change in this number of epochs will cause
   * stagnation.
   */
  unsigned int cascade_output_stagnation_epochs;

  /* The error must change by at least this
   * fraction of its old value to count as a
   * significant change.
   */
  float cascade_candidate_change_fraction;

  /* No change in this number of epochs will cause
   * stagnation.
   */
  unsigned int cascade_candidate_stagnation_epochs;

  /* The current best candidate, which will be installed.
   */
  unsigned int cascade_best_candidate;

  /* The upper limit for a candidate score
   */
  fann_type cascade_candidate_limit;

  /* Scale of copied candidate output weights
   */
  fann_type cascade_weight_multiplier;

  /* Maximum epochs to train the output neurons during cascade training
   */
  unsigned int cascade_max_out_epochs;

  /* Maximum epochs to train the candidate neurons during cascade training
   */
  unsigned int cascade_max_cand_epochs;

  /* Minimum epochs to train the output neurons during cascade training
   */
  unsigned int cascade_min_out_epochs;

  /* Minimum epochs to train the candidate neurons during cascade training
   */
  unsigned int cascade_min_cand_epochs;

  /* An array consisting of the activation functions used when doing
   * cascade training.
   */
  enum fann_activationfunc_enum *cascade_activation_functions;

  /* The number of elements in the cascade_activation_functions array.
   */
  unsigned int cascade_activation_functions_count;

  /* An array consisting of the steepnesses used during cascade training.
   */
  fann_type *cascade_activation_steepnesses;

  /* The number of elements in the cascade_activation_steepnesses array.
   */
  unsigned int cascade_activation_steepnesses_count;

  /* The number of candidates of each type that will be present.
   * The actual number of candidates is then
   * cascade_activation_functions_count *
   * cascade_activation_steepnesses_count *
   * cascade_num_candidate_groups
   */
  unsigned int cascade_num_candidate_groups;

  /* An array consisting of the score of the individual candidates,
   * which is used to decide which candidate is the best
   */
  fann_type *cascade_candidate_scores;

  /* The number of allocated neurons during cascade correlation algorithms.
   * This number might be higher than the actual number of neurons to avoid
   * allocating new space too often.
   */
  unsigned int total_neurons_allocated;

  /* The number of allocated connections during cascade correlation algorithms.
   * This number might be higher than the actual number of neurons to avoid
   * allocating new space too often.
   */
  unsigned int total_connections_allocated;

  /* Variables for use with Quickprop training */

  /* Decay is used to make the weights not go so high */
  float quickprop_decay;

  /* Mu is a factor used to increase and decrease the stepsize */
  float quickprop_mu;

  /* Variables for use with with RPROP training */

  /* Tells how much the stepsize should increase during learning */
  float rprop_increase_factor;

  /* Tells how much the stepsize should decrease during learning */
  float rprop_decrease_factor;

  /* The minimum stepsize */
  float rprop_delta_min;

  /* The maximum stepsize */
  float rprop_delta_max;

  /* The initial stepsize */
  float rprop_delta_zero;

  /* Defines how much the weights are constrained to smaller values at the beginning */
  float sarprop_weight_decay_shift;

  /* Decides if the stepsize is too big with regard to the error */
  float sarprop_step_error_threshold_factor;

  /* Defines how much the stepsize is influenced by the error */
  float sarprop_step_error_shift;

  /* Defines how much the epoch influences weight decay and noise */
  float sarprop_temperature;

  /* Current training epoch */
  unsigned int sarprop_epoch;

  /* Used to contain the slope errors used during batch training
   * Is allocated during first training session,
   * which means that if we do not train, it is never allocated.
   */
  fann_type *train_slopes;

  /* The previous step taken by the quickprop/rprop procedures.
   * Not allocated if not used.
   */
  fann_type *prev_steps;

  /* The slope values used by the quickprop/rprop procedures.
   * Not allocated if not used.
   */
  fann_type *prev_train_slopes;

  /* The last delta applied to a connection weight.
   * This is used for the momentum term in the backpropagation algorithm.
   * Not allocated if not used.
   */
  fann_type *prev_weights_deltas;

};

/* Type: fann_connection
    Describes a connection between two neurons and its weight
    from_neuron - Unique number used to identify source neuron
    to_neuron - Unique number used to identify destination neuron
    weight - The numerical value of the weight
    See Also:
        <fann_get_connection_array>, <fann_set_weight_array>
   This structure appears in FANN >= 2.1.0
*/
struct fann_connection {
  /* Unique number used to identify source neuron */
  unsigned int from_neuron;
  /* Unique number used to identify destination neuron */
  unsigned int to_neuron;
  /* The numerical value of the weight */
  fann_type weight;
};

float FANN_API fann_get_MSE(struct fann *ann);

fann_type fann_update_MSE(struct fann *ann, struct fann_neuron *neuron, fann_type neuron_diff);

struct fann *fann_allocate_structure(unsigned int num_layers);
void fann_allocate_neurons(struct fann *ann);

void fann_allocate_connections(struct fann *ann);

int fann_save_internal(struct fann *ann, const char *configuration_file,
                       unsigned int save_as_fixed);
int fann_save_internal_fd(struct fann *ann, FILE *conf, const char *configuration_file,
                          unsigned int save_as_fixed);
int fann_save_train_internal(struct fann_train_data *data, const char *filename,
                             unsigned int save_as_fixed, unsigned int decimal_point);
int fann_save_train_internal_fd(struct fann_train_data *data, FILE *file, const char *filename,
                                unsigned int save_as_fixed, unsigned int decimal_point);

void fann_update_stepwise(struct fann *ann);
void fann_seed_rand();

void fann_error(struct fann_error *errdat, const enum fann_errno_enum errno_f, ...);
void fann_init_error_data(struct fann_error *errdat);

struct fann *fann_create_from_fd(FILE *conf, const char *configuration_file);
struct fann_train_data *fann_read_train_from_fd(FILE *file, const char *filename);

void fann_compute_MSE(struct fann *ann, fann_type *desired_output);
void fann_update_output_weights(struct fann *ann);
void fann_backpropagate_MSE(struct fann *ann);
void fann_update_weights(struct fann *ann);
void fann_update_slopes_batch(struct fann *ann, struct fann_layer *layer_begin,
                              struct fann_layer *layer_end);
void fann_update_weights_quickprop(struct fann *ann, unsigned int num_data,
                                   unsigned int first_weight, unsigned int past_end);
void fann_update_weights_batch(struct fann *ann, unsigned int num_data, unsigned int first_weight,
                               unsigned int past_end);
void fann_update_weights_irpropm(struct fann *ann, unsigned int first_weight,
                                 unsigned int past_end);
void fann_update_weights_sarprop(struct fann *ann, unsigned int epoch, unsigned int first_weight,
                                 unsigned int past_end);

void fann_clear_train_arrays(struct fann *ann);

fann_type fann_activation(struct fann *ann, unsigned int activation_function, fann_type steepness,
                          fann_type value);

fann_type fann_activation_derived(unsigned int activation_function, fann_type steepness,
                                  fann_type value, fann_type sum);

int fann_desired_error_reached(struct fann *ann, float desired_error);

/* Some functions for cascade */
int fann_train_outputs(struct fann *ann, struct fann_train_data *data, float desired_error);

float fann_train_outputs_epoch(struct fann *ann, struct fann_train_data *data);

int fann_train_candidates(struct fann *ann, struct fann_train_data *data);

fann_type fann_train_candidates_epoch(struct fann *ann, struct fann_train_data *data);

void fann_install_candidate(struct fann *ann);
int fann_check_input_output_sizes(struct fann *ann, struct fann_train_data *data);

int fann_initialize_candidates(struct fann *ann);

void fann_set_shortcut_connections(struct fann *ann);

int fann_allocate_scale(struct fann *ann);

__global__ void Add_Mult_update_wts(fann_type* A, fann_type B, fann_type* C, const float constant, int N, fann_neuron* D);

void FANN_API fann_scale_data_to_range(fann_type **data, unsigned int num_data,
                                                     unsigned int num_elem, fann_type old_min,
                                                     fann_type old_max, fann_type new_min,
                                                     fann_type new_max);

void fann_compute_MSE(struct fann *ann, fann_type *desired_output) {
  fann_type neuron_value, neuron_diff, *error_it = 0, *error_begin = 0;
  struct fann_neuron *last_layer_begin = (ann->last_layer - 1)->first_neuron;
  const struct fann_neuron *last_layer_end = last_layer_begin + ann->num_output;
  const struct fann_neuron *first_neuron = ann->first_layer->first_neuron;

  /* if no room allocated for the error variabels, allocate it now */
  if (ann->train_errors == NULL) {
    ann->train_errors = (fann_type *)calloc(ann->total_neurons, sizeof(fann_type));
    if (ann->train_errors == NULL) {
      fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
      return;
    }
  } else {
    /* clear the error variabels */
    memset(ann->train_errors, 0, (ann->total_neurons) * sizeof(fann_type));
  }
  error_begin = ann->train_errors;

  /* calculate the error and place it in the output layer */
  error_it = error_begin + (last_layer_begin - first_neuron);

  for (; last_layer_begin != last_layer_end; last_layer_begin++) {
    neuron_value = last_layer_begin->value;
    neuron_diff = *desired_output - neuron_value;

    neuron_diff = fann_update_MSE(ann, last_layer_begin, neuron_diff);

    if (ann->train_error_function) { /* TODO make switch when more functions */
      if (neuron_diff < -.9999999)
        neuron_diff = -17.0;
      else if (neuron_diff > .9999999)
        neuron_diff = 17.0;
      else
        neuron_diff = (fann_type)log((1.0 + neuron_diff) / (1.0 - neuron_diff));
    }

    *error_it = fann_activation_derived(last_layer_begin->activation_function,
                                        last_layer_begin->activation_steepness, neuron_value,
                                        last_layer_begin->sum) *
                neuron_diff;

    desired_output++;
    error_it++;

    ann->num_MSE++;
  }
}

fann_type fann_activation_derived(unsigned int activation_function, fann_type steepness,
                                  fann_type value, fann_type sum) {
  switch (activation_function) {
    case FANN_LINEAR:
    case FANN_LINEAR_PIECE:
    case FANN_LINEAR_PIECE_SYMMETRIC:
      return (fann_type)fann_linear_derive(steepness, value);
    case FANN_SIGMOID:
    case FANN_SIGMOID_STEPWISE:
      value = fann_clip(value, 0.01f, 0.99f);
      return (fann_type)fann_sigmoid_derive(steepness, value);
    case FANN_SIGMOID_SYMMETRIC:
    case FANN_SIGMOID_SYMMETRIC_STEPWISE:
      value = fann_clip(value, -0.98f, 0.98f);
      return (fann_type)fann_sigmoid_symmetric_derive(steepness, value);
    case FANN_GAUSSIAN:
      /* value = fann_clip(value, 0.01f, 0.99f); */
      return (fann_type)fann_gaussian_derive(steepness, value, sum);
    case FANN_GAUSSIAN_SYMMETRIC:
      /* value = fann_clip(value, -0.98f, 0.98f); */
      return (fann_type)fann_gaussian_symmetric_derive(steepness, value, sum);
    case FANN_ELLIOT:
      value = fann_clip(value, 0.01f, 0.99f);
      return (fann_type)fann_elliot_derive(steepness, value, sum);
    case FANN_ELLIOT_SYMMETRIC:
      value = fann_clip(value, -0.98f, 0.98f);
      return (fann_type)fann_elliot_symmetric_derive(steepness, value, sum);
    case FANN_SIN_SYMMETRIC:
      return (fann_type)fann_sin_symmetric_derive(steepness, sum);
    case FANN_COS_SYMMETRIC:
      return (fann_type)fann_cos_symmetric_derive(steepness, sum);
    case FANN_SIN:
      return (fann_type)fann_sin_derive(steepness, sum);
    case FANN_COS:
      return (fann_type)fann_cos_derive(steepness, sum);
    case FANN_THRESHOLD:
      fann_error(NULL, FANN_E_CANT_TRAIN_ACTIVATION);
  }
  return 0;
}
/* INTERNAL FUNCTION
   Propagate the error backwards from the output layer.
   After this the train_errors in the hidden layers will be:
   neuron_value_derived * sum(outgoing_weights * connected_neuron)
*/
__global__ void backpropagate_gpu(int N, fann_type* output, fann_type temp, fann_type* input)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i <= N)
    {
      output[N-i]+=temp*input[N-i];
    }
}


void fann_backpropagate_MSE(struct fann *ann) {
  
  fann_type tmp_error;
  unsigned int i;
  struct fann_layer *layer_it;
  struct fann_neuron *neuron_it, *last_neuron;
  struct fann_neuron **connections;

  fann_type *error_begin = ann->train_errors;
  fann_type *error_prev_layer;
  fann_type *weights;
  const struct fann_neuron *first_neuron = ann->first_layer->first_neuron;
  const struct fann_layer *second_layer = ann->first_layer + 1;
  struct fann_layer *last_layer = ann->last_layer;
  
  // threads, blocks

  int threadsperblock;
  int numBlocks;
  int N;

  //device parameters
  // device parameters
  fann_type* d_errors;
  fann_type* d_wts;

  threadsperblock=256;
  
  /* go through all the layers, from last to first.
   * And propagate the error backwards */
  for (layer_it = last_layer - 1; layer_it > second_layer; --layer_it) {
    last_neuron = layer_it->last_neuron;

    /* for each connection in this layer, propagate the error backwards */
    if (ann->connection_rate >= 1) {
      if (ann->network_type == FANN_NETTYPE_LAYER) {
        error_prev_layer = error_begin + ((layer_it - 1)->first_neuron - first_neuron);
      } else {
        error_prev_layer = error_begin;
      }
      
      
      for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
        tmp_error = error_begin[neuron_it - first_neuron];
        weights = ann->weights + neuron_it->first_con;

        N=neuron_it->last_con - neuron_it->first_con;
        hipMalloc(&d_errors,N*sizeof(fann_type));
      hipMalloc(&d_wts,N*sizeof(fann_type));
        
      hipMemcpy(d_errors,error_prev_layer,N*sizeof(fann_type),hipMemcpyHostToDevice);
      hipMemcpy(d_wts,weights,N*sizeof(fann_type),hipMemcpyHostToDevice);
        

        numBlocks=(N+threadsperblock-1)/threadsperblock;
        
        backpropagate_gpu<<<numBlocks,threadsperblock>>>(N,d_errors,tmp_error,d_wts);

        hipMemcpy(error_prev_layer,d_errors,N*sizeof(fann_type),hipMemcpyDeviceToHost);
        hipMemcpy(weights,d_wts,N*sizeof(fann_type),hipMemcpyDeviceToHost);

        hipFree(d_errors);
        hipFree(d_wts);
      }
      

    } else {
      for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
        tmp_error = error_begin[neuron_it - first_neuron];
        weights = ann->weights + neuron_it->first_con;
        connections = ann->connections + neuron_it->first_con;
        for (i = neuron_it->last_con - neuron_it->first_con; i--;) {
          error_begin[connections[i] - first_neuron] += tmp_error * weights[i];
        }
      }
    }

    /* then calculate the actual errors in the previous layer */
    error_prev_layer = error_begin + ((layer_it - 1)->first_neuron - first_neuron);
    last_neuron = (layer_it - 1)->last_neuron;

    for (neuron_it = (layer_it - 1)->first_neuron; neuron_it != last_neuron; neuron_it++) {
      *error_prev_layer *=
          fann_activation_derived(neuron_it->activation_function, neuron_it->activation_steepness,
                                  neuron_it->value, neuron_it->sum);
      error_prev_layer++;
    }
  }
}

/* INTERNAL FUNCTION
   Update weights for incremental training
*/
// Device code
__global__ void Add_Mult_update_wts(fann_type* A, fann_type B, fann_type* C, const float constant, int N, fann_neuron* D)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
      fann_type delta_w = B * D[i].value + constant * A[i];
      C[i] += delta_w;
      A[i] = delta_w;
    }
}

void fann_update_weights(struct fann *ann) {
  
  struct fann_neuron *neuron_it, *last_neuron, *prev_neurons;
  fann_type tmp_error, *weights;
  struct fann_layer *layer_it;
  
  unsigned int num_connections;

  /* store some variabels local for fast access */
  const float learning_rate = ann->learning_rate;
  const float learning_momentum = ann->learning_momentum;
  struct fann_neuron *first_neuron = ann->first_layer->first_neuron;
  struct fann_layer *first_layer = ann->first_layer;
  const struct fann_layer *last_layer = ann->last_layer;
  fann_type *error_begin = ann->train_errors;
  fann_type *deltas_begin, *weights_deltas;

  // device parameters
  fann_type* d_wts;
  fann_type* d_wt_del;
  fann_neuron* d_prev_neurons;
  
  int threadsperblock=256;

  /* if no room allocated for the deltas, allocate it now */
  if (ann->prev_weights_deltas == NULL) {
    ann->prev_weights_deltas =
        (fann_type *)calloc(ann->total_connections_allocated, sizeof(fann_type));
    if (ann->prev_weights_deltas == NULL) {
      fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
      return;
    }
  }

  deltas_begin = ann->prev_weights_deltas;
  prev_neurons = first_neuron;
  for (layer_it = (first_layer + 1); layer_it != last_layer; layer_it++) {

    last_neuron = layer_it->last_neuron;
    if (ann->connection_rate >= 1) {
      if (ann->network_type == FANN_NETTYPE_LAYER) {
        prev_neurons = (layer_it - 1)->first_neuron;
      }
      for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
        tmp_error = error_begin[neuron_it - first_neuron] * learning_rate;
        num_connections = neuron_it->last_con - neuron_it->first_con;
        weights = ann->weights + neuron_it->first_con;
        weights_deltas = deltas_begin + neuron_it->first_con;

        hipMalloc(&d_wts,num_connections*sizeof(fann_type));
        hipMalloc(&d_wt_del,num_connections*sizeof(fann_type));

        hipMemcpy(d_wts,weights,num_connections*sizeof(fann_type),hipMemcpyHostToDevice);
        hipMemcpy(d_wt_del,weights_deltas,num_connections*sizeof(fann_type),hipMemcpyHostToDevice);

        hipMalloc(&d_prev_neurons,num_connections*sizeof(fann_neuron));
        hipMemcpy(d_prev_neurons,prev_neurons,num_connections*sizeof(fann_neuron),hipMemcpyHostToDevice);
        
        int numblocks=num_connections/threadsperblock;
        
        Add_Mult_update_wts<<<numblocks,threadsperblock>>>(d_wt_del,tmp_error,d_wts,learning_momentum,num_connections,d_prev_neurons);

        hipMemcpy(weights,d_wts,num_connections*sizeof(fann_type),hipMemcpyDeviceToHost);
        hipMemcpy(weights_deltas,d_wt_del,num_connections*sizeof(fann_type),hipMemcpyDeviceToHost);
        hipMemcpy(prev_neurons,d_prev_neurons,num_connections*sizeof(fann_neuron),hipMemcpyDeviceToHost);
        }
      }
     else {
      for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
        tmp_error = error_begin[neuron_it - first_neuron] * learning_rate;
        num_connections = neuron_it->last_con - neuron_it->first_con;
        weights = ann->weights + neuron_it->first_con;
        weights_deltas = deltas_begin + neuron_it->first_con;
        
        hipMalloc(&d_wts,num_connections*sizeof(fann_type));
        hipMalloc(&d_wt_del,num_connections*sizeof(fann_type));

        hipMemcpy(d_wts,weights,num_connections*sizeof(fann_type),hipMemcpyHostToDevice);
        hipMemcpy(d_wt_del,weights_deltas,num_connections*sizeof(fann_type),hipMemcpyHostToDevice);

        hipMalloc(&d_prev_neurons,num_connections*sizeof(fann_neuron));
        hipMemcpy(d_prev_neurons,prev_neurons,num_connections*sizeof(fann_neuron),hipMemcpyHostToDevice);
        
        
        int numblocks=num_connections/threadsperblock;
        Add_Mult_update_wts<<<numblocks,threadsperblock>>>(d_wt_del,tmp_error,d_wts,learning_momentum,num_connections,d_prev_neurons);
        
        hipMemcpy(weights,d_wts,num_connections*sizeof(fann_type),hipMemcpyDeviceToHost);
        hipMemcpy(weights_deltas,d_wt_del,num_connections*sizeof(fann_type),hipMemcpyDeviceToHost);
        hipMemcpy(prev_neurons,d_prev_neurons,num_connections*sizeof(fann_neuron),hipMemcpyDeviceToHost);

        hipFree(d_wts);
        hipFree(d_wt_del);
        hipFree(d_prev_neurons);
      }
    }
  }
}



/* INTERNAL FUNCTION
   Update slopes for batch training
   layer_begin = ann->first_layer+1 and layer_end = ann->last_layer-1
   will update all slopes.
*/

__global__ void slopes_gpu(int N, fann_type* nrn_slope, fann_type tmp, fann_neuron* val)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
      nrn_slope[i]+=tmp*val[i].value;
    }

}

__global__ void slopes_gpu2(int N, fann_type* nrn_slope, fann_type tmp, fann_neuron** val)
{ 
  int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
      nrn_slope[i]+=tmp*val[i]->value;
    }

}

void fann_update_slopes_batch(struct fann *ann, struct fann_layer *layer_begin,
                              struct fann_layer *layer_end) {
  struct fann_neuron *neuron_it, *last_neuron, *prev_neurons, **connections;
  fann_type tmp_error;
  unsigned int num_connections;

  /* store some variabels local for fast access */
  struct fann_neuron *first_neuron = ann->first_layer->first_neuron;
  fann_type *error_begin = ann->train_errors;
  fann_type *slope_begin, *neuron_slope;

  /* if no room allocated for the slope variabels, allocate it now */
  if (ann->train_slopes == NULL) {
    ann->train_slopes = (fann_type *)calloc(ann->total_connections_allocated, sizeof(fann_type));
    if (ann->train_slopes == NULL) {
      fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
      return;
    }
  }

  if (layer_begin == NULL) {
    layer_begin = ann->first_layer + 1;
  }

  if (layer_end == NULL) {
    layer_end = ann->last_layer - 1;
  }

  slope_begin = ann->train_slopes;

  prev_neurons = first_neuron;

  // device parameters
  fann_type* d_nrn_slope;
  fann_neuron* d_prev_nrn;
  fann_neuron** d_connect;
  int threadsperblock=256;
  int numBlocks;

  for (; layer_begin <= layer_end; layer_begin++) {

    last_neuron = layer_begin->last_neuron;
    if (ann->connection_rate >= 1) {
      if (ann->network_type == FANN_NETTYPE_LAYER) {
        prev_neurons = (layer_begin - 1)->first_neuron;
      }

      for (neuron_it = layer_begin->first_neuron; neuron_it != last_neuron; neuron_it++) {
        tmp_error = error_begin[neuron_it - first_neuron];
        neuron_slope = slope_begin + neuron_it->first_con;
        num_connections = neuron_it->last_con - neuron_it->first_con;

        hipMalloc(&d_nrn_slope,num_connections*sizeof(fann_type));
        hipMalloc(&d_prev_nrn,num_connections*sizeof(fann_neuron));

        hipMemcpy(d_nrn_slope,neuron_slope,num_connections*sizeof(fann_type),hipMemcpyHostToDevice);
        hipMemcpy(d_prev_nrn,prev_neurons,num_connections*sizeof(fann_neuron),hipMemcpyHostToDevice);
        
        numBlocks=(num_connections+threadsperblock-1)/threadsperblock;

        slopes_gpu<<<numBlocks,threadsperblock>>>(num_connections,d_nrn_slope,tmp_error,d_prev_nrn);

        hipMemcpy(neuron_slope,d_nrn_slope,num_connections*sizeof(fann_type),hipMemcpyDeviceToHost);
        hipMemcpy(prev_neurons,d_prev_nrn,num_connections*sizeof(fann_neuron),hipMemcpyDeviceToHost);

        hipFree(d_nrn_slope);
        hipFree(d_prev_nrn);
      }
    } else {
      for (neuron_it = layer_begin->first_neuron; neuron_it != last_neuron; neuron_it++) {
        tmp_error = error_begin[neuron_it - first_neuron];
        neuron_slope = slope_begin + neuron_it->first_con;
        num_connections = neuron_it->last_con - neuron_it->first_con;
        connections = ann->connections + neuron_it->first_con;

        hipMalloc(&d_nrn_slope,num_connections*sizeof(fann_type));
        hipMalloc(&d_connect,num_connections*sizeof(fann_neuron));

        hipMemcpy(d_nrn_slope,neuron_slope,num_connections*sizeof(fann_type),hipMemcpyHostToDevice);
        hipMemcpy(d_connect,connections,num_connections*sizeof(fann_neuron),hipMemcpyHostToDevice);
        
        numBlocks=(num_connections+threadsperblock-1)/threadsperblock;

        slopes_gpu2<<<numBlocks,threadsperblock>>>(num_connections,d_nrn_slope,tmp_error,d_connect);

        hipMemcpy(neuron_slope,d_nrn_slope,num_connections*sizeof(fann_type),hipMemcpyDeviceToHost);
        hipMemcpy(connections,d_connect,num_connections*sizeof(fann_neuron),hipMemcpyDeviceToHost);

        hipFree(d_nrn_slope);
        hipFree(d_connect);

      }
    }
  }
}

/* INTERNAL FUNCTION
   Clears arrays used for training before a new training session.
   Also creates the arrays that do not exist yet.
 */
void fann_clear_train_arrays(struct fann *ann) {
  unsigned int i;
  fann_type delta_zero;

  /* if no room allocated for the slope variabels, allocate it now
   * (calloc clears mem) */
  if (ann->train_slopes == NULL) {
    ann->train_slopes = (fann_type *)calloc(ann->total_connections_allocated, sizeof(fann_type));
    if (ann->train_slopes == NULL) {
      fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
      return;
    }
  } else {
    memset(ann->train_slopes, 0, (ann->total_connections_allocated) * sizeof(fann_type));
  }

  /* if no room allocated for the variabels, allocate it now */
  if (ann->prev_steps == NULL) {
    ann->prev_steps = (fann_type *)malloc(ann->total_connections_allocated * sizeof(fann_type));
    if (ann->prev_steps == NULL) {
      fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
      return;
    }
  }

  if (ann->training_algorithm == FANN_TRAIN_RPROP) {
    delta_zero = ann->rprop_delta_zero;

    for (i = 0; i < ann->total_connections_allocated; i++) ann->prev_steps[i] = delta_zero;
  } else {
    memset(ann->prev_steps, 0, (ann->total_connections_allocated) * sizeof(fann_type));
  }

  /* if no room allocated for the variabels, allocate it now */
  if (ann->prev_train_slopes == NULL) {
    ann->prev_train_slopes =
        (fann_type *)calloc(ann->total_connections_allocated, sizeof(fann_type));
    if (ann->prev_train_slopes == NULL) {
      fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
      return;
    }
  } else {
    memset(ann->prev_train_slopes, 0, (ann->total_connections_allocated) * sizeof(fann_type));
  }
}

/* INTERNAL FUNCTION
   Update weights for batch training
 */
void fann_update_weights_batch(struct fann *ann, unsigned int num_data, unsigned int first_weight,
                               unsigned int past_end) {
  fann_type *train_slopes = ann->train_slopes;
  fann_type *weights = ann->weights;
  const float epsilon = ann->learning_rate / num_data;
  unsigned int i = first_weight;

  for (; i != past_end; i++) {
    weights[i] += train_slopes[i] * epsilon;
    train_slopes[i] = 0.0;
  }
}

/* INTERNAL FUNCTION
   The quickprop training algorithm
 */

void fann_update_weights_quickprop(struct fann *ann, unsigned int num_data,
                                   unsigned int first_weight, unsigned int past_end) {
  fann_type *train_slopes = ann->train_slopes;
  fann_type *weights = ann->weights;
  fann_type *prev_steps = ann->prev_steps;
  fann_type *prev_train_slopes = ann->prev_train_slopes;

  fann_type w, prev_step, slope, prev_slope, next_step;

  float epsilon = ann->learning_rate / num_data;
  float decay = ann->quickprop_decay; /*-0.0001;*/
  float mu = ann->quickprop_mu;       /*1.75; */
  float shrink_factor = (float)(mu / (1.0 + mu));

  unsigned int i = first_weight;

  for (; i != past_end; i++) {
    w = weights[i];
    prev_step = prev_steps[i];
    slope = train_slopes[i] + decay * w;
    prev_slope = prev_train_slopes[i];
    next_step = 0.0;

    /* The step must always be in direction opposite to the slope. */
    if (prev_step > 0.001) {
      /* If last step was positive...  */
      if (slope > 0.0) /*  Add in linear term if current slope is still positive. */
        next_step += epsilon * slope;

      /*If current slope is close to or larger than prev slope...  */
      if (slope > (shrink_factor * prev_slope))
        next_step += mu * prev_step; /* Take maximum size negative step. */
      else
        next_step += prev_step * slope / (prev_slope - slope); /* Else, use quadratic estimate. */
    } else if (prev_step < -0.001) {
      /* If last step was negative...  */
      if (slope < 0.0) /*  Add in linear term if current slope is still negative. */
        next_step += epsilon * slope;

      /* If current slope is close to or more neg than prev slope... */
      if (slope < (shrink_factor * prev_slope))
        next_step += mu * prev_step; /* Take maximum size negative step. */
      else
        next_step += prev_step * slope / (prev_slope - slope); /* Else, use quadratic estimate. */
    } else /* Last step was zero, so use only linear term. */
      next_step += epsilon * slope;

    /*
    if(next_step > 1000 || next_step < -1000)
    {
            printf("quickprop[%d] weight=%f, slope=%f, prev_slope=%f, next_step=%f, prev_step=%f\n",
                       i, weights[i], slope, prev_slope, next_step, prev_step);
               if(next_step > 1000)
               next_step = 1000;
               else
               next_step = -1000;
    }
*/

    /* update global data arrays */
    prev_steps[i] = next_step;

    w += next_step;

    if (w > 1500)
      weights[i] = 1500;
    else if (w < -1500)
      weights[i] = -1500;
    else
      weights[i] = w;

    /*weights[i] = w;*/

    prev_train_slopes[i] = slope;
    train_slopes[i] = 0.0;
  }
}

/* INTERNAL FUNCTION
   The iRprop- algorithm
*/
__global__ void gpu_update_wts(int begin, int end, fann_type* train_slopes, fann_type* prev_train_slopes, fann_type* prev_steps,
float increase_factor, float delta_max, float decrease_factor, float delta_min, fann_type* weights)
{
fann_type prev_step, slope, prev_slope, next_step, same_sign;
int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i>=begin & i<end)
{
prev_step = fann_max(
        prev_steps[i],
        (fann_type)0.0001); /* prev_step may not be zero because then the training will stop */
    slope = train_slopes[i];
    prev_slope = prev_train_slopes[i];

    same_sign = prev_slope * slope;

    if (same_sign >= 0.0)
      next_step = fann_min(prev_step * increase_factor, delta_max);
    else {
      next_step = fann_max(prev_step * decrease_factor, delta_min);
      slope = 0;
    }

    if (slope < 0) {
      weights[i] -= next_step;
      if (weights[i] < -1500) weights[i] = -1500;
    } else {
      weights[i] += next_step;
      if (weights[i] > 1500) weights[i] = 1500;
    }

    /*if(i == 2){
     * printf("weight=%f, slope=%f, next_step=%f, prev_step=%f\n", weights[i], slope, next_step,
     * prev_step);
     * } */

    /* update global data arrays */
    prev_steps[i] = next_step;
    prev_train_slopes[i] = slope;
    train_slopes[i] = 0.0;
}
}

void fann_update_weights_irpropm(struct fann *ann, unsigned int first_weight,
                                 unsigned int past_end) {
  fann_type *train_slopes = ann->train_slopes;
  fann_type *weights = ann->weights;
  fann_type *prev_steps = ann->prev_steps;
  fann_type *prev_train_slopes = ann->prev_train_slopes;

  float increase_factor = ann->rprop_increase_factor; /*1.2; */
  float decrease_factor = ann->rprop_decrease_factor; /*0.5; */
  float delta_min = ann->rprop_delta_min;             /*0.0; */
  float delta_max = ann->rprop_delta_max;             /*50.0; */

  unsigned int i = first_weight;

  // device parameters
  fann_type* d_train_slopes;
  fann_type* d_prev_train_slopes; 
  fann_type* d_prev_steps;
  fann_type* d_weights;
  
  int N=past_end-first_weight;

  hipMalloc(&d_train_slopes,N*sizeof(fann_type));
  hipMalloc(&d_prev_train_slopes,N*sizeof(fann_type));
  hipMalloc(&d_prev_steps,N*sizeof(fann_type));
  hipMalloc(&d_weights,N*sizeof(fann_type));

  hipMemcpy(d_train_slopes,train_slopes,N*sizeof(fann_type),hipMemcpyHostToDevice);
  hipMemcpy(d_prev_train_slopes,prev_train_slopes,N*sizeof(fann_type),hipMemcpyHostToDevice);
  hipMemcpy(d_prev_steps,prev_steps,N*sizeof(fann_type),hipMemcpyHostToDevice);
  hipMemcpy(d_weights,weights,N*sizeof(fann_type),hipMemcpyHostToDevice);

  int threadsperblock=256;
  int numBlocks=(N+threadsperblock-1)/threadsperblock;

  gpu_update_wts<<<numBlocks,threadsperblock>>>(i,past_end,d_train_slopes,d_prev_train_slopes,d_prev_steps,increase_factor,delta_max,decrease_factor,delta_min,d_weights);
  
  hipMemcpy(train_slopes,d_train_slopes,N*sizeof(fann_type),hipMemcpyDeviceToHost);
  hipMemcpy(prev_train_slopes,d_prev_train_slopes,N*sizeof(fann_type),hipMemcpyDeviceToHost);
  hipMemcpy(prev_steps,d_prev_steps,N*sizeof(fann_type),hipMemcpyDeviceToHost);
  hipMemcpy(weights,d_weights,N*sizeof(fann_type),hipMemcpyDeviceToHost);

  hipFree(d_train_slopes);
  hipFree(d_prev_steps);
  hipFree(d_prev_train_slopes);
  hipFree(d_weights);

}

/* INTERNAL FUNCTION
   The SARprop- algorithm
*/
void fann_update_weights_sarprop(struct fann *ann, unsigned int epoch, unsigned int first_weight,
                                 unsigned int past_end) {
  fann_type *train_slopes = ann->train_slopes;
  fann_type *weights = ann->weights;
  fann_type *prev_steps = ann->prev_steps;
  fann_type *prev_train_slopes = ann->prev_train_slopes;

  fann_type prev_step, slope, prev_slope, next_step = 0, same_sign;

  /* These should be set from variables */
  float increase_factor = ann->rprop_increase_factor; /*1.2; */
  float decrease_factor = ann->rprop_decrease_factor; /*0.5; */
  /* TODO: why is delta_min 0.0 in iRprop? SARPROP uses 1x10^-6 (Braun and Riedmiller, 1993) */
  float delta_min = 0.000001f;
  float delta_max = ann->rprop_delta_max;                     /*50.0; */
  float weight_decay_shift = ann->sarprop_weight_decay_shift; /* ld 0.01 = -6.644 */
  float step_error_threshold_factor = ann->sarprop_step_error_threshold_factor; /* 0.1 */
  float step_error_shift = ann->sarprop_step_error_shift;                       /* ld 3 = 1.585 */
  float T = ann->sarprop_temperature;
  float MSE = fann_get_MSE(ann);
  float RMSE = sqrtf(MSE);

  unsigned int i = first_weight;

  /* for all weights; TODO: are biases included? */
  for (; i != past_end; i++) {
    /* TODO: confirm whether 1x10^-6 == delta_min is really better */
    prev_step = fann_max(
        prev_steps[i],
        (fann_type)0.000001); /* prev_step may not be zero because then the training will stop */
    /* calculate SARPROP slope; TODO: better as new error function? (see SARPROP paper)*/
    slope = -train_slopes[i] - weights[i] * (fann_type)fann_exp2(-T * epoch + weight_decay_shift);

    /* TODO: is prev_train_slopes[i] 0.0 in the beginning? */
    prev_slope = prev_train_slopes[i];

    same_sign = prev_slope * slope;

    if (same_sign > 0.0) {
      next_step = fann_min(prev_step * increase_factor, delta_max);
      /* TODO: are the signs inverted? see differences between SARPROP paper and iRprop */
      if (slope < 0.0)
        weights[i] += next_step;
      else
        weights[i] -= next_step;
    } else if (same_sign < 0.0) {
      if (prev_step < step_error_threshold_factor * MSE)
        next_step =
            prev_step * decrease_factor +
            (float)rand() / RAND_MAX * RMSE * (fann_type)fann_exp2(-T * epoch + step_error_shift);
      else
        next_step = fann_max(prev_step * decrease_factor, delta_min);

      slope = 0.0;
    } else {
      if (slope < 0.0)
        weights[i] += prev_step;
      else
        weights[i] -= prev_step;
    }

    /*if(i == 2){
     * printf("weight=%f, slope=%f, next_step=%f, prev_step=%f\n", weights[i], slope, next_step,
     * prev_step);
     * } */

    /* update global data arrays */
    prev_steps[i] = next_step;
    prev_train_slopes[i] = slope;
    train_slopes[i] = 0.0;
  }
}

   struct fann *FANN_API fann_create_standard_array(unsigned int num_layers,
                                                               const unsigned int *layers);

/* Function: fann_create_sparse
        Creates a standard backpropagation neural network, which is not fully connected.
        Parameters:
                connection_rate - The connection rate controls how many connections there will be in
   the network. If the connection rate is set to 1, the network will be fully connected, but if it
   is set to 0.5 only half of the connections will be set. A connection rate of 1 will yield the
   same result as <fann_create_standard> num_layers - The total number of layers including the input
   and the output layer.
                ... - Integer values determining the number of neurons in each layer starting with
   the input layer and ending with the output layer.
        Returns:
                A pointer to the newly created <struct fann>.
        See also:
                <fann_create_sparse_array>, <fann_create_standard>, <fann_create_shortcut>
        This function appears in FANN >= 2.0.0.
*/
   struct fann *FANN_API fann_create_sparse(float connection_rate,
                                                       unsigned int num_layers, ...);

/* Function: fann_create_sparse_array
   Just like <fann_create_sparse>, but with an array of layer sizes
   instead of individual parameters.
        See <fann_create_standard_array> for a description of the parameters.
        See also:
                <fann_create_sparse>, <fann_create_standard>, <fann_create_shortcut>
        This function appears in FANN >= 2.0.0.
*/
   struct fann *FANN_API fann_create_sparse_array(float connection_rate,
                                                             unsigned int num_layers,
                                                             const unsigned int *layers);

/* Function: fann_create_shortcut
        Creates a standard backpropagation neural network, which is fully connected and which
        also has shortcut connections.
        Shortcut connections are connections that skip layers. A fully connected network with
   shortcut connections is a network where all neurons are connected to all neurons in later layers.
        Including direct connections from the input layer to the output layer.
        See <fann_create_standard> for a description of the parameters.
        See also:
                <fann_create_shortcut_array>, <fann_create_standard>, <fann_create_sparse>,
        This function appears in FANN >= 2.0.0.
*/
   struct fann *FANN_API fann_create_shortcut(unsigned int num_layers, ...);

/* Function: fann_create_shortcut_array
   Just like <fann_create_shortcut>, but with an array of layer sizes
   instead of individual parameters.
        See <fann_create_standard_array> for a description of the parameters.
        See also:
                <fann_create_shortcut>, <fann_create_standard>, <fann_create_sparse>
        This function appears in FANN >= 2.0.0.
*/
   struct fann *FANN_API fann_create_shortcut_array(unsigned int num_layers,
                                                               const unsigned int *layers);
/* Function: fann_destroy
   Destroys the entire network, properly freeing all the associated memory.
        This function appears in FANN >= 1.0.0.
*/
   void FANN_API fann_destroy(struct fann *ann);

/* Function: fann_copy
   Creates a copy of a fann structure.
   Data in the user data <fann_set_user_data> is not copied, but the user data pointer is copied.
        This function appears in FANN >= 2.2.0.
*/
   struct fann *FANN_API fann_copy(struct fann *ann);

/* Function: fann_run
        Will run input through the neural network, returning an array of outputs, the number of
   which being equal to the number of neurons in the output layer.
        See also:
                <fann_test>
        This function appears in FANN >= 1.0.0.
*/
   fann_type *FANN_API fann_run(struct fann *ann, fann_type *input);

/* Function: fann_randomize_weights
        Give each connection a random weight between *min_weight* and *max_weight*
        From the beginning the weights are random between -0.1 and 0.1.
        See also:
                <fann_init_weights>
        This function appears in FANN >= 1.0.0.
*/
   void FANN_API fann_randomize_weights(struct fann *ann, fann_type min_weight,
                                                   fann_type max_weight);

/* Function: fann_init_weights
        Initialize the weights using Widrow + Nguyen's algorithm.
        This function behaves similarly to fann_randomize_weights. It will use the algorithm
   developed by Derrick Nguyen and Bernard Widrow to set the weights in such a way as to speed up
   training. This technique is not always successful, and in some cases can be less efficient than a
   purely random initialization.
        The algorithm requires access to the range of the input data (ie, largest and smallest
   input), and therefore accepts a second argument, data, which is the training data that will be
   used to train the network.
        See also:
                <fann_randomize_weights>, <fann_read_train_from_file>
        This function appears in FANN >= 1.1.0.
*/
   void FANN_API fann_init_weights(struct fann *ann, struct fann_train_data *train_data);

/* Function: fann_print_connections
        Will print the connections of the ann in a compact matrix, for easy viewing of the internals
        of the ann.
        The output from fann_print_connections on a small (2 2 1) network trained on the xor problem
        >Layer / Neuron 012345
        >L   1 / N    3 BBa...
        >L   1 / N    4 BBA...
        >L   1 / N    5 ......
        >L   2 / N    6 ...BBA
        >L   2 / N    7 ......
        This network has five real neurons and two bias neurons. This gives a total of seven neurons
        named from 0 to 6. The connections between these neurons can be seen in the matrix. "." is a
        place where there is no connection, while a character tells how strong the connection is on
   a scale from a-z. The two real neurons in the hidden layer (neuron 3 and 4 in layer 1) have
        connections from the three neurons in the previous layer as is visible in the first two
   lines. The output neuron (6) has connections from the three neurons in the hidden layer 3 - 5 as
   is visible in the fourth line.
        To simplify the matrix output neurons are not visible as neurons that connections can come
   from, and input and bias neurons are not visible as neurons that connections can go to.
        This function appears in FANN >= 1.2.0.
*/
   void FANN_API fann_print_connections(struct fann *ann);

/* Group: Parameters */
/* Function: fann_print_parameters
        Prints all of the parameters and options of the ANN
        This function appears in FANN >= 1.2.0.
*/
   void FANN_API fann_print_parameters(struct fann *ann);

/* Function: fann_get_num_input
   Get the number of input neurons.
        This function appears in FANN >= 1.0.0.
*/
   unsigned int FANN_API fann_get_num_input(struct fann *ann);

/* Function: fann_get_num_output
   Get the number of output neurons.
        This function appears in FANN >= 1.0.0.
*/
   unsigned int FANN_API fann_get_num_output(struct fann *ann);

/* Function: fann_get_total_neurons
   Get the total number of neurons in the entire network. This number does also include the
        bias neurons, so a 2-4-2 network has 2+4+2 +2(bias) = 10 neurons.
        This function appears in FANN >= 1.0.0.
*/
   unsigned int FANN_API fann_get_total_neurons(struct fann *ann);

/* Function: fann_get_total_connections
   Get the total number of connections in the entire network.
        This function appears in FANN >= 1.0.0.
*/
   unsigned int FANN_API fann_get_total_connections(struct fann *ann);

/* Function: fann_get_network_type
    Get the type of neural network it was created as.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
        Returns:
        The neural network type from enum <fann_network_type_enum>
    See Also:
        <fann_network_type_enum>
   This function appears in FANN >= 2.1.0
*/
   enum fann_nettype_enum FANN_API fann_get_network_type(struct fann *ann);

/* Function: fann_get_connection_rate
    Get the connection rate used when the network was created
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
        Returns:
        The connection rate
   This function appears in FANN >= 2.1.0
*/
   float FANN_API fann_get_connection_rate(struct fann *ann);

/* Function: fann_get_num_layers
    Get the number of layers in the network
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
        Returns:
                The number of layers in the neural network
        Example:
                > // Obtain the number of layers in a neural network
                > struct fann *ann = fann_create_standard(4, 2, 8, 9, 1);
        > unsigned int num_layers = fann_get_num_layers(ann);
   This function appears in FANN >= 2.1.0
*/
   unsigned int FANN_API fann_get_num_layers(struct fann *ann);

/*Function: fann_get_layer_array
    Get the number of neurons in each layer in the network.
    Bias is not included so the layers match the fann_create functions.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
    The layers array must be preallocated to at least
    sizeof(unsigned int) * fann_num_layers() long.
   This function appears in FANN >= 2.1.0
*/
   void FANN_API fann_get_layer_array(struct fann *ann, unsigned int *layers);

/* Function: fann_get_bias_array
    Get the number of bias in each layer in the network.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
    The bias array must be preallocated to at least
    sizeof(unsigned int) * fann_num_layers() long.
   This function appears in FANN >= 2.1.0
*/
   void FANN_API fann_get_bias_array(struct fann *ann, unsigned int *bias);

/* Function: fann_get_connection_array
    Get the connections in the network.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
    The connections array must be preallocated to at least
    sizeof(struct fann_connection) * fann_get_total_connections() long.
   This function appears in FANN >= 2.1.0
*/
   void FANN_API fann_get_connection_array(struct fann *ann,
                                                      struct fann_connection *connections);

/* Function: fann_set_weight_array
    Set connections in the network.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
    Only the weights can be changed, connections and weights are ignored
    if they do not already exist in the network.
    The array must have sizeof(struct fann_connection) * num_connections size.
   This function appears in FANN >= 2.1.0
*/
   void FANN_API fann_set_weight_array(struct fann *ann,
                                                  struct fann_connection *connections,
                                                  unsigned int num_connections);

/* Function: fann_set_weight
    Set a connection in the network.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
    Only the weights can be changed. The connection/weight is
    ignored if it does not already exist in the network.
   This function appears in FANN >= 2.1.0
*/
   void FANN_API fann_set_weight(struct fann *ann, unsigned int from_neuron,
                                            unsigned int to_neuron, fann_type weight);

/* Function: fann_get_weights
    Get all the network weights.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
                weights - A fann_type pointer to user data. It is the responsibility
                        of the user to allocate sufficient space to store all the weights.
   This function appears in FANN >= x.y.z
*/
   void FANN_API fann_get_weights(struct fann *ann, fann_type *weights);

/* Function: fann_set_weights
    Set network weights.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
                weights - A fann_type pointer to user data. It is the responsibility
                        of the user to make the weights array sufficient long
                        to store all the weights.
   This function appears in FANN >= x.y.z
*/
   void FANN_API fann_set_weights(struct fann *ann, fann_type *weights);

/* Function: fann_set_user_data
    Store a pointer to user defined data. The pointer can be
    retrieved with <fann_get_user_data> for example in a
    callback. It is the user's responsibility to allocate and
    deallocate any data that the pointer might point to.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
                user_data - A void pointer to user defined data.
   This function appears in FANN >= 2.1.0
*/
   void FANN_API fann_set_user_data(struct fann *ann, void *user_data);

/* Function: fann_get_user_data
    Get a pointer to user defined data that was previously set
    with <fann_set_user_data>. It is the user's responsibility to
    allocate and deallocate any data that the pointer might point to.
    Parameters:
                ann - A previously created neural network structure of
            type <struct fann> pointer.
    Returns:
        A void pointer to user defined data.
   This function appears in FANN >= 2.1.0
*/
   void *FANN_API fann_get_user_data(struct fann *ann);

/* Function: fann_disable_seed_rand
   Disables the automatic random generator seeding that happens in FANN.
   Per default FANN will always seed the random generator when creating a new network,
   unless FANN_NO_SEED is defined during compilation of the library. This method can
   disable this at runtime.
   This function appears in FANN >= 2.3.0
*/
   void FANN_API fann_disable_seed_rand();

/* Function: fann_enable_seed_rand
   Enables the automatic random generator seeding that happens in FANN.
   Per default FANN will always seed the random generator when creating a new network,
   unless FANN_NO_SEED is defined during compilation of the library. This method can
   disable this at runtime.
   This function appears in FANN >= 2.3.0
*/
   void FANN_API fann_enable_seed_rand();

unsigned int FANN_API fann_get_num_output(struct fann *ann)
{
  return ann->num_output;
}

struct fann *FANN_API fann_create_standard(unsigned int num_layers, ...) {
  struct fann *ann;
  va_list layer_sizes;
  int i;
  int status;
  int arg;
  unsigned int *layers = (unsigned int *)calloc(num_layers, sizeof(unsigned int));

  if (layers == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  va_start(layer_sizes, num_layers);

  status = 1;
  for (i = 0; i < (int)num_layers; i++) {
    arg = va_arg(layer_sizes, unsigned int);
    if (arg < 0 || arg > 1000000) status = 0;
    layers[i] = arg;
  }
  va_end(layer_sizes);

  if (!status) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    free(layers);
    return NULL;
  }

  ann = fann_create_standard_array(num_layers, layers);

  free(layers);

  return ann;
}

 struct fann *FANN_API fann_create_standard_array(unsigned int num_layers,
                                                               const unsigned int *layers) {
  return fann_create_sparse_array(1, num_layers, layers);
}

  struct fann *FANN_API fann_create_sparse(float connection_rate,
                                                       unsigned int num_layers, ...) {
  struct fann *ann;
  va_list layer_sizes;
  int i;
  int status;
  int arg;
  unsigned int *layers = (unsigned int *)calloc(num_layers, sizeof(unsigned int));

  va_start(layer_sizes, num_layers);
  status = 1;
  for (i = 0; i < (int)num_layers; i++) {
    arg = va_arg(layer_sizes, unsigned int);
    if (arg < 0 || arg > 1000000) status = 0;
    layers[i] = arg;
  }
  va_end(layer_sizes);

  if (!status) {
    
    free(layers);
    return NULL;
  }

  ann = fann_create_sparse_array(connection_rate, num_layers, layers);
  free(layers);

  return ann;
}

  struct fann *FANN_API fann_create_sparse_array(float connection_rate,
                                                             unsigned int num_layers,
                                                             const unsigned int *layers) {
  struct fann_layer *layer_it, *last_layer, *prev_layer;
  struct fann *ann;
  struct fann_neuron *neuron_it, *last_neuron, *random_neuron, *bias_neuron;

  unsigned int num_neurons_in, num_neurons_out, i, j;
  unsigned int min_connections, max_connections, num_connections;
  unsigned int connections_per_neuron, allocated_connections;
  unsigned int random_number, found_connection, tmp_con;

  if (connection_rate > 1) {
    connection_rate = 1;
  }

  fann_seed_rand();

  /* allocate the general structure */
  ann = fann_allocate_structure(num_layers);
  if (ann == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  ann->connection_rate = connection_rate;

  /* determine how many neurons there should be in each layer */
  i = 0;
  for (layer_it = ann->first_layer; layer_it != ann->last_layer; layer_it++) {
    /* we do not allocate room here, but we make sure that
     * last_neuron - first_neuron is the number of neurons */
    layer_it->first_neuron = NULL;
    layer_it->last_neuron = layer_it->first_neuron + layers[i++] + 1; /* +1 for bias */
    ann->total_neurons += (unsigned int)(layer_it->last_neuron - layer_it->first_neuron);
  }

  ann->num_output =
      (unsigned int)((ann->last_layer - 1)->last_neuron - (ann->last_layer - 1)->first_neuron - 1);
  ann->num_input =
      (unsigned int)(ann->first_layer->last_neuron - ann->first_layer->first_neuron - 1);

  /* allocate room for the actual neurons */
  fann_allocate_neurons(ann);
  if (ann->errno_f == FANN_E_CANT_ALLOCATE_MEM) {
    fann_destroy(ann);
    return NULL;
  }


  num_neurons_in = ann->num_input;
  for (layer_it = ann->first_layer + 1; layer_it != ann->last_layer; layer_it++) {
    num_neurons_out = (unsigned int)(layer_it->last_neuron - layer_it->first_neuron - 1);
    /*if all neurons in each layer should be connected to at least one neuron
     * in the previous layer, and one neuron in the next layer.
     * and the bias node should be connected to the all neurons in the next layer.
     * Then this is the minimum amount of neurons */
    min_connections = fann_max(num_neurons_in, num_neurons_out); /* not calculating bias */
    max_connections = num_neurons_in * num_neurons_out;          /* not calculating bias */
    num_connections =
        fann_max(min_connections, (unsigned int)(0.5 + (connection_rate * max_connections))) +
        num_neurons_out;

    connections_per_neuron = num_connections / num_neurons_out;
    allocated_connections = 0;
    /* Now split out the connections on the different neurons */
    for (i = 0; i != num_neurons_out; i++) {
      layer_it->first_neuron[i].first_con = ann->total_connections + allocated_connections;
      allocated_connections += connections_per_neuron;
      layer_it->first_neuron[i].last_con = ann->total_connections + allocated_connections;

      layer_it->first_neuron[i].activation_function = FANN_SIGMOID_STEPWISE;

      layer_it->first_neuron[i].activation_steepness = 0.5;


      if (allocated_connections < (num_connections * (i + 1)) / num_neurons_out) {
        layer_it->first_neuron[i].last_con++;
        allocated_connections++;
      }
    }

    /* bias neuron also gets stuff */
    layer_it->first_neuron[i].first_con = ann->total_connections + allocated_connections;
    layer_it->first_neuron[i].last_con = ann->total_connections + allocated_connections;

    ann->total_connections += num_connections;

    /* used in the next run of the loop */
    num_neurons_in = num_neurons_out;
  }

  fann_allocate_connections(ann);
  if (ann->errno_f == FANN_E_CANT_ALLOCATE_MEM) {
    fann_destroy(ann);
    return NULL;
  }

  if (connection_rate >= 1) {

    prev_layer = ann->first_layer;
    last_layer = ann->last_layer;
    for (layer_it = ann->first_layer + 1; layer_it != last_layer; layer_it++) {
      last_neuron = layer_it->last_neuron - 1;
      for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
        tmp_con = neuron_it->last_con - 1;
        for (i = neuron_it->first_con; i != tmp_con; i++) {
          ann->weights[i] = (fann_type)fann_random_weight();
          /* these connections are still initialized for fully connected networks, to allow
           * operations to work, that are not optimized for fully connected networks.
           */
          ann->connections[i] = prev_layer->first_neuron + (i - neuron_it->first_con);
        }

        /* bias weight */
        ann->weights[tmp_con] = (fann_type)fann_random_bias_weight();
        ann->connections[tmp_con] = prev_layer->first_neuron + (tmp_con - neuron_it->first_con);
      }

    }
  } else {
    /* make connections for a network, that are not fully connected */

    /* generally, what we do is first to connect all the input
     * neurons to a output neuron, respecting the number of
     * available input neurons for each output neuron. Then
     * we go through all the output neurons, and connect the
     * rest of the connections to input neurons, that they are
     * not allready connected to.
     */

    /* All the connections are cleared by calloc, because we want to
     * be able to see which connections are allready connected */

    for (layer_it = ann->first_layer + 1; layer_it != ann->last_layer; layer_it++) {
      num_neurons_out = (unsigned int)(layer_it->last_neuron - layer_it->first_neuron - 1);
      num_neurons_in =
          (unsigned int)((layer_it - 1)->last_neuron - (layer_it - 1)->first_neuron - 1);

      /* first connect the bias neuron */
      bias_neuron = (layer_it - 1)->last_neuron - 1;
      last_neuron = layer_it->last_neuron - 1;
      for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
        ann->connections[neuron_it->first_con] = bias_neuron;
        ann->weights[neuron_it->first_con] = (fann_type)fann_random_bias_weight();
      }

      /* then connect all neurons in the input layer */
      last_neuron = (layer_it - 1)->last_neuron - 1;
      for (neuron_it = (layer_it - 1)->first_neuron; neuron_it != last_neuron; neuron_it++) {
        /* random neuron in the output layer that has space
         * for more connections */
        do {
          random_number = (int)(0.5 + fann_rand(0, num_neurons_out - 1));
          random_neuron = layer_it->first_neuron + random_number;
          /* checks the last space in the connections array for room */
        } while (ann->connections[random_neuron->last_con - 1]);

        /* find an empty space in the connection array and connect */
        for (i = random_neuron->first_con; i < random_neuron->last_con; i++) {
          if (ann->connections[i] == NULL) {
            ann->connections[i] = neuron_it;
            ann->weights[i] = (fann_type)fann_random_weight();
            break;
          }
        }
      }

      /* then connect the rest of the unconnected neurons */
      last_neuron = layer_it->last_neuron - 1;
      for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
        /* find empty space in the connection array and connect */
        for (i = neuron_it->first_con; i < neuron_it->last_con; i++) {
          /* continue if allready connected */
          if (ann->connections[i] != NULL) continue;

          do {
            found_connection = 0;
            random_number = (int)(0.5 + fann_rand(0, num_neurons_in - 1));
            random_neuron = (layer_it - 1)->first_neuron + random_number;

            /* check to see if this connection is allready there */
            for (j = neuron_it->first_con; j < i; j++) {
              if (random_neuron == ann->connections[j]) {
                found_connection = 1;
                break;
              }
            }

          } while (found_connection);

          /* we have found a neuron that is not allready
           * connected to us, connect it */
          ann->connections[i] = random_neuron;
          ann->weights[i] = (fann_type)fann_random_weight();
        }
      }


    }

    /* TODO it would be nice to have the randomly created
     * connections sorted for smoother memory access.
     */
  }



  return ann;
}

  struct fann *FANN_API fann_create_shortcut(unsigned int num_layers, ...) {
  struct fann *ann;
  int i;
  int status;
  int arg;
  va_list layer_sizes;
  unsigned int *layers = (unsigned int *)calloc(num_layers, sizeof(unsigned int));

  if (layers == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  va_start(layer_sizes, num_layers);
  status = 1;
  for (i = 0; i < (int)num_layers; i++) {
    arg = va_arg(layer_sizes, unsigned int);
    if (arg < 0 || arg > 1000000) status = 0;
    layers[i] = arg;
  }
  va_end(layer_sizes);

  if (!status) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    free(layers);
    return NULL;
  }

  ann = fann_create_shortcut_array(num_layers, layers);

  free(layers);

  return ann;
}

  struct fann *FANN_API fann_create_shortcut_array(unsigned int num_layers,
                                                               const unsigned int *layers) {
  struct fann_layer *layer_it, *layer_it2, *last_layer;
  struct fann *ann;
  struct fann_neuron *neuron_it, *neuron_it2 = 0;
  unsigned int i;
  unsigned int num_neurons_in, num_neurons_out;

  fann_seed_rand();

  /* allocate the general structure */
  ann = fann_allocate_structure(num_layers);
  if (ann == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  ann->connection_rate = 1;
  ann->network_type = FANN_NETTYPE_SHORTCUT;

  /* determine how many neurons there should be in each layer */
  i = 0;
  for (layer_it = ann->first_layer; layer_it != ann->last_layer; layer_it++) {
    /* we do not allocate room here, but we make sure that
     * last_neuron - first_neuron is the number of neurons */
    layer_it->first_neuron = NULL;
    layer_it->last_neuron = layer_it->first_neuron + layers[i++];
    if (layer_it == ann->first_layer) {
      /* there is a bias neuron in the first layer */
      layer_it->last_neuron++;
    }

    ann->total_neurons += (unsigned int)(layer_it->last_neuron - layer_it->first_neuron);
  }

  ann->num_output =
      (unsigned int)((ann->last_layer - 1)->last_neuron - (ann->last_layer - 1)->first_neuron);
  ann->num_input =
      (unsigned int)(ann->first_layer->last_neuron - ann->first_layer->first_neuron - 1);

  /* allocate room for the actual neurons */
  fann_allocate_neurons(ann);
  if (ann->errno_f == FANN_E_CANT_ALLOCATE_MEM) {
    fann_destroy(ann);
    return NULL;
  }



  num_neurons_in = ann->num_input;
  last_layer = ann->last_layer;
  for (layer_it = ann->first_layer + 1; layer_it != last_layer; layer_it++) {
    num_neurons_out = (unsigned int)(layer_it->last_neuron - layer_it->first_neuron);

    /* Now split out the connections on the different neurons */
    for (i = 0; i != num_neurons_out; i++) {
      layer_it->first_neuron[i].first_con = ann->total_connections;
      ann->total_connections += num_neurons_in + 1;
      layer_it->first_neuron[i].last_con = ann->total_connections;

      layer_it->first_neuron[i].activation_function = FANN_SIGMOID_STEPWISE;

      layer_it->first_neuron[i].activation_steepness = 0.5;

    }


    /* used in the next run of the loop */
    num_neurons_in += num_neurons_out;
  }

  fann_allocate_connections(ann);
  if (ann->errno_f == FANN_E_CANT_ALLOCATE_MEM) {
    fann_destroy(ann);
    return NULL;
  }

  /* Connections are created from all neurons to all neurons in later layers
   */
  num_neurons_in = ann->num_input + 1;
  for (layer_it = ann->first_layer + 1; layer_it != last_layer; layer_it++) {
    for (neuron_it = layer_it->first_neuron; neuron_it != layer_it->last_neuron; neuron_it++) {
      i = neuron_it->first_con;
      for (layer_it2 = ann->first_layer; layer_it2 != layer_it; layer_it2++) {
        for (neuron_it2 = layer_it2->first_neuron; neuron_it2 != layer_it2->last_neuron;
             neuron_it2++) {
          ann->weights[i] = (fann_type)fann_random_weight();
          ann->connections[i] = neuron_it2;
          i++;
        }
      }
    }
    num_neurons_in += (unsigned int)(layer_it->last_neuron - layer_it->first_neuron);
  }



  return ann;
}

__global__ void assign_fann_run(fann_type* out,fann_neuron* nrn, int N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
      out[i]=nrn[i].value;
    }

}

__global__ void assign2_fann_run(fann_neuron* out,fann_type* nrn, int N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
      out[i].value=nrn[i];
    }

}

  fann_type *FANN_API fann_run(struct fann *ann, fann_type *input) {
  struct fann_neuron *neuron_it, *last_neuron, *neurons, **neuron_pointers;
  unsigned int i, num_connections, num_input, num_output;
  fann_type neuron_sum, *output;
  fann_type *weights;
  struct fann_layer *layer_it, *last_layer;
  unsigned int activation_function;
  fann_type steepness;
  
  int threadsperblock, numBlocks;
  /* store some variabels local for fast access */
  struct fann_neuron *first_neuron = ann->first_layer->first_neuron;

  fann_type max_sum = 0;
  
  //device parameters
  fann_neuron* d_first_neuron;
  fann_type* d_input;


  /* first set the input */
  num_input = ann->num_input;

  hipMalloc(&d_first_neuron,num_input*sizeof(fann_neuron));
  hipMalloc(&d_input,num_input*sizeof(fann_type));

  hipMemcpy(d_first_neuron,first_neuron,num_input*sizeof(fann_neuron),hipMemcpyHostToDevice);
  hipMemcpy(d_input,input,num_input*sizeof(fann_type),hipMemcpyHostToDevice);
  
  threadsperblock=256;
  numBlocks = (num_input + threadsperblock - 1) / threadsperblock;
  dim3 dimBlock(threadsperblock);
  dim3 dimGrid(numBlocks);

  assign2_fann_run<<<dimGrid,dimBlock>>>(d_first_neuron,d_input,num_input);
  
  //cudaDeviceSynchronize();

  hipMemcpy(first_neuron,d_first_neuron,num_input*sizeof(fann_neuron),hipMemcpyDeviceToHost);
  hipMemcpy(input,d_input,num_input*sizeof(fann_type),hipMemcpyDeviceToHost);
  
  hipFree(d_first_neuron);
  hipFree(d_input);

  /* Set the bias neuron in the input layer */

  (ann->first_layer->last_neuron - 1)->value = 1;


  last_layer = ann->last_layer;
  for (layer_it = ann->first_layer + 1; layer_it != last_layer; layer_it++) {
    last_neuron = layer_it->last_neuron;
    for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
      if (neuron_it->first_con == neuron_it->last_con) {
        /* bias neurons */

        neuron_it->value = 1;

        continue;
      }

      activation_function = neuron_it->activation_function;
      steepness = neuron_it->activation_steepness;

      neuron_sum = 0;
      num_connections = neuron_it->last_con - neuron_it->first_con;
      weights = ann->weights + neuron_it->first_con;

      if (ann->connection_rate >= 1) {
        if (ann->network_type == FANN_NETTYPE_SHORTCUT) {
          neurons = ann->first_layer->first_neuron;
        } else {
          neurons = (layer_it - 1)->first_neuron;
        }

        /* unrolled loop start */
        i = num_connections & 3; /* same as modulo 4 */
        switch (i) {
          case 3:
            neuron_sum += fann_mult(weights[2], neurons[2].value);
          case 2:
            neuron_sum += fann_mult(weights[1], neurons[1].value);
          case 1:
            neuron_sum += fann_mult(weights[0], neurons[0].value);
          case 0:
            break;
        }

        for (; i != num_connections; i += 4) {
          neuron_sum += fann_mult(weights[i], neurons[i].value) +
                        fann_mult(weights[i + 1], neurons[i + 1].value) +
                        fann_mult(weights[i + 2], neurons[i + 2].value) +
                        fann_mult(weights[i + 3], neurons[i + 3].value);
        }
        /* unrolled loop end */

        /*
         * for(i = 0;i != num_connections; i++){
         * printf("%f += %f*%f, ", neuron_sum, weights[i], neurons[i].value);
         * neuron_sum += fann_mult(weights[i], neurons[i].value);
         * }
         */
      } else {
        neuron_pointers = ann->connections + neuron_it->first_con;

        i = num_connections & 3; /* same as modulo 4 */
        switch (i) {
          case 3:
            neuron_sum += fann_mult(weights[2], neuron_pointers[2]->value);
          case 2:
            neuron_sum += fann_mult(weights[1], neuron_pointers[1]->value);
          case 1:
            neuron_sum += fann_mult(weights[0], neuron_pointers[0]->value);
          case 0:
            break;
        }

        for (; i != num_connections; i += 4) {
          neuron_sum += fann_mult(weights[i], neuron_pointers[i]->value) +
                        fann_mult(weights[i + 1], neuron_pointers[i + 1]->value) +
                        fann_mult(weights[i + 2], neuron_pointers[i + 2]->value) +
                        fann_mult(weights[i + 3], neuron_pointers[i + 3]->value);
        }
      }


      neuron_sum = fann_mult(steepness, neuron_sum);

      max_sum = 150 / steepness;
      if (neuron_sum > max_sum)
        neuron_sum = max_sum;
      else if (neuron_sum < -max_sum)
        neuron_sum = -max_sum;

      neuron_it->sum = neuron_sum;

      fann_activation_switch(activation_function, neuron_sum, neuron_it->value);

    }
  }

  /* set the output */
  output = ann->output;
  num_output = ann->num_output;
  neurons = (ann->last_layer - 1)->first_neuron;

  // device parameters
  fann_type* d_output;
  fann_neuron* d_neuron;

  hipMalloc(&d_output,num_output*sizeof(fann_type));
  hipMalloc(&d_neuron,num_output*sizeof(fann_neuron));

  hipMemcpy(d_output,output,num_output*sizeof(fann_type),hipMemcpyHostToDevice);
  hipMemcpy(d_neuron,neurons,num_output*sizeof(fann_neuron),hipMemcpyHostToDevice);
  
  threadsperblock=256;
  numBlocks = (num_output + threadsperblock - 1) / threadsperblock;
  dim3 dimBlock2(threadsperblock);
  dim3 dimGrid2(numBlocks);

  assign_fann_run<<<dimGrid2,dimBlock2>>>(d_output,d_neuron,num_output);
  
  //cudaDeviceSynchronize();

  hipMemcpy(output,d_output,num_output*sizeof(fann_type),hipMemcpyDeviceToHost);
  hipMemcpy(neurons,d_neuron,num_output*sizeof(fann_neuron),hipMemcpyDeviceToHost);
  
  hipFree(d_output);
  hipFree(d_neuron);
 
  return ann->output;

}



  void FANN_API fann_destroy(struct fann *ann) {
  if (ann == NULL) return;
  fann_safe_free(ann->weights);
  fann_safe_free(ann->connections);
  fann_safe_free(ann->first_layer->first_neuron);
  fann_safe_free(ann->first_layer);
  fann_safe_free(ann->output);
  fann_safe_free(ann->train_errors);
  fann_safe_free(ann->train_slopes);
  fann_safe_free(ann->prev_train_slopes);
  fann_safe_free(ann->prev_steps);
  fann_safe_free(ann->prev_weights_deltas);
  fann_safe_free(ann->errstr);
  fann_safe_free(ann->cascade_activation_functions);
  fann_safe_free(ann->cascade_activation_steepnesses);
  fann_safe_free(ann->cascade_candidate_scores);



  fann_safe_free(ann);
}

  void FANN_API fann_randomize_weights(struct fann *ann, fann_type min_weight,
                                                   fann_type max_weight) {
  fann_type *last_weight;
  fann_type *weights = ann->weights;

  last_weight = weights + ann->total_connections;
  for (; weights != last_weight; weights++) {
    *weights = (fann_type)(fann_rand(min_weight, max_weight));
  }


}

/* deep copy of the fann structure */
  struct fann *FANN_API fann_copy(struct fann *orig) {
  struct fann *copy;
  unsigned int num_layers = (unsigned int)(orig->last_layer - orig->first_layer);
  struct fann_layer *orig_layer_it, *copy_layer_it;
  unsigned int layer_size;
  struct fann_neuron *last_neuron, *orig_neuron_it, *copy_neuron_it;
  unsigned int i;
  struct fann_neuron *orig_first_neuron, *copy_first_neuron;
  unsigned int input_neuron;

  copy = fann_allocate_structure(num_layers);
  if (copy == NULL) {
    fann_error((struct fann_error *)orig, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }
  copy->errno_f = orig->errno_f;
  if (orig->errstr) {
    copy->errstr = (char *)malloc(FANN_ERRSTR_MAX);
    if (copy->errstr == NULL) {
      fann_destroy(copy);
      return NULL;
    }
    strcpy(copy->errstr, orig->errstr);
  }
  copy->error_log = orig->error_log;

  copy->learning_rate = orig->learning_rate;
  copy->learning_momentum = orig->learning_momentum;
  copy->connection_rate = orig->connection_rate;
  copy->network_type = orig->network_type;
  copy->num_MSE = orig->num_MSE;
  copy->MSE_value = orig->MSE_value;
  copy->num_bit_fail = orig->num_bit_fail;
  copy->bit_fail_limit = orig->bit_fail_limit;
  copy->train_error_function = orig->train_error_function;
  copy->train_stop_function = orig->train_stop_function;
  copy->training_algorithm = orig->training_algorithm;
  copy->callback = orig->callback;
  copy->user_data = orig->user_data;

  copy->quickprop_decay = orig->quickprop_decay;
  copy->quickprop_mu = orig->quickprop_mu;
  copy->rprop_increase_factor = orig->rprop_increase_factor;
  copy->rprop_decrease_factor = orig->rprop_decrease_factor;
  copy->rprop_delta_min = orig->rprop_delta_min;
  copy->rprop_delta_max = orig->rprop_delta_max;
  copy->rprop_delta_zero = orig->rprop_delta_zero;

  /* user_data is not deep copied.  user should use fann_copy_with_user_data() for that */
  copy->user_data = orig->user_data;



  /* copy layer sizes, prepare for fann_allocate_neurons */
  for (orig_layer_it = orig->first_layer, copy_layer_it = copy->first_layer;
       orig_layer_it != orig->last_layer; orig_layer_it++, copy_layer_it++) {
    layer_size = (unsigned int)(orig_layer_it->last_neuron - orig_layer_it->first_neuron);
    copy_layer_it->first_neuron = NULL;
    copy_layer_it->last_neuron = copy_layer_it->first_neuron + layer_size;
    copy->total_neurons += layer_size;
  }
  copy->num_input = orig->num_input;
  copy->num_output = orig->num_output;

  /* copy scale parameters, when used */


  /* copy the neurons */
  fann_allocate_neurons(copy);
  if (copy->errno_f == FANN_E_CANT_ALLOCATE_MEM) {
    fann_destroy(copy);
    return NULL;
  }
  layer_size =
      (unsigned int)((orig->last_layer - 1)->last_neuron - (orig->last_layer - 1)->first_neuron);
  memcpy(copy->output, orig->output, layer_size * sizeof(fann_type));

  last_neuron = (orig->last_layer - 1)->last_neuron;
  for (orig_neuron_it = orig->first_layer->first_neuron,
      copy_neuron_it = copy->first_layer->first_neuron;
       orig_neuron_it != last_neuron; orig_neuron_it++, copy_neuron_it++) {
    memcpy(copy_neuron_it, orig_neuron_it, sizeof(struct fann_neuron));
  }
  /* copy the connections */
  copy->total_connections = orig->total_connections;
  fann_allocate_connections(copy);
  if (copy->errno_f == FANN_E_CANT_ALLOCATE_MEM) {
    fann_destroy(copy);
    return NULL;
  }

  orig_first_neuron = orig->first_layer->first_neuron;
  copy_first_neuron = copy->first_layer->first_neuron;
  for (i = 0; i < orig->total_connections; i++) {
    copy->weights[i] = orig->weights[i];
    input_neuron = (unsigned int)(orig->connections[i] - orig_first_neuron);
    copy->connections[i] = copy_first_neuron + input_neuron;
  }

  if (orig->train_slopes) {
    copy->train_slopes = (fann_type *)malloc(copy->total_connections_allocated * sizeof(fann_type));
    if (copy->train_slopes == NULL) {
      fann_error((struct fann_error *)orig, FANN_E_CANT_ALLOCATE_MEM);
      fann_destroy(copy);
      return NULL;
    }
    memcpy(copy->train_slopes, orig->train_slopes,
           copy->total_connections_allocated * sizeof(fann_type));
  }

  if (orig->prev_steps) {
    copy->prev_steps = (fann_type *)malloc(copy->total_connections_allocated * sizeof(fann_type));
    if (copy->prev_steps == NULL) {
      fann_error((struct fann_error *)orig, FANN_E_CANT_ALLOCATE_MEM);
      fann_destroy(copy);
      return NULL;
    }
    memcpy(copy->prev_steps, orig->prev_steps,
           copy->total_connections_allocated * sizeof(fann_type));
  }

  if (orig->prev_train_slopes) {
    copy->prev_train_slopes =
        (fann_type *)malloc(copy->total_connections_allocated * sizeof(fann_type));
    if (copy->prev_train_slopes == NULL) {
      fann_error((struct fann_error *)orig, FANN_E_CANT_ALLOCATE_MEM);
      fann_destroy(copy);
      return NULL;
    }
    memcpy(copy->prev_train_slopes, orig->prev_train_slopes,
           copy->total_connections_allocated * sizeof(fann_type));
  }

  if (orig->prev_weights_deltas) {
    copy->prev_weights_deltas =
        (fann_type *)malloc(copy->total_connections_allocated * sizeof(fann_type));
    if (copy->prev_weights_deltas == NULL) {
      fann_error((struct fann_error *)orig, FANN_E_CANT_ALLOCATE_MEM);
      fann_destroy(copy);
      return NULL;
    }
    memcpy(copy->prev_weights_deltas, orig->prev_weights_deltas,
           copy->total_connections_allocated * sizeof(fann_type));
  }

  return copy;
}

  void FANN_API fann_print_connections(struct fann *ann) {
  struct fann_layer *layer_it;
  struct fann_neuron *neuron_it;
  unsigned int i;
  int value;
  char *neurons;
  unsigned int num_neurons = fann_get_total_neurons(ann) - fann_get_num_output(ann);

  neurons = (char *)malloc(num_neurons + 1);
  if (neurons == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    return;
  }
  neurons[num_neurons] = 0;

  printf("Layer / Neuron ");
  for (i = 0; i < num_neurons; i++) {
    printf("%d", i % 10);
  }
  printf("\n");

  for (layer_it = ann->first_layer + 1; layer_it != ann->last_layer; layer_it++) {
    for (neuron_it = layer_it->first_neuron; neuron_it != layer_it->last_neuron; neuron_it++) {
      memset(neurons, (int)'.', num_neurons);
      for (i = neuron_it->first_con; i < neuron_it->last_con; i++) {
        if (ann->weights[i] < 0) {

          value = (int)((ann->weights[i]) - 0.5);

          if (value < -25) value = -25;
          neurons[ann->connections[i] - ann->first_layer->first_neuron] = (char)('a' - value);
        } else {

          value = (int)((ann->weights[i]) + 0.5);

          if (value > 25) value = 25;
          neurons[ann->connections[i] - ann->first_layer->first_neuron] = (char)('A' + value);
        }
      }
      printf("L %3d / N %4d %s\n", (int)(layer_it - ann->first_layer),
             (int)(neuron_it - ann->first_layer->first_neuron), neurons);
    }
  }

  free(neurons);
}

/* Initialize the weights using Widrow + Nguyen's algorithm.
 */
  void FANN_API fann_init_weights(struct fann *ann,
                                              struct fann_train_data *train_data) {
  fann_type smallest_inp, largest_inp;
  unsigned int dat = 0, elem, num_connect, num_hidden_neurons;
  struct fann_layer *layer_it;
  struct fann_neuron *neuron_it, *last_neuron, *bias_neuron;


  float scale_factor;

  for (smallest_inp = largest_inp = train_data->input[0][0]; dat < train_data->num_data; dat++) {
    for (elem = 0; elem < train_data->num_input; elem++) {
      if (train_data->input[dat][elem] < smallest_inp) smallest_inp = train_data->input[dat][elem];
      if (train_data->input[dat][elem] > largest_inp) largest_inp = train_data->input[dat][elem];
    }
  }

  num_hidden_neurons = (unsigned int)(ann->total_neurons - (ann->num_input + ann->num_output +
                                                            (ann->last_layer - ann->first_layer)));
  scale_factor = (float)(pow((double)(0.7f * (double)num_hidden_neurons),
                             (double)(1.0f / (double)ann->num_input)) /
                         (double)(largest_inp - smallest_inp));


  bias_neuron = ann->first_layer->last_neuron - 1;
  for (layer_it = ann->first_layer + 1; layer_it != ann->last_layer; layer_it++) {
    last_neuron = layer_it->last_neuron;

    if (ann->network_type == FANN_NETTYPE_LAYER) {
      bias_neuron = (layer_it - 1)->last_neuron - 1;
    }

    for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
      for (num_connect = neuron_it->first_con; num_connect < neuron_it->last_con; num_connect++) {
        if (bias_neuron == ann->connections[num_connect]) {

          ann->weights[num_connect] = (fann_type)fann_rand(-scale_factor, scale_factor);

        } else {

          ann->weights[num_connect] = (fann_type)fann_rand(0, scale_factor);

        }
      }
    }
  }


}

  void FANN_API fann_print_parameters(struct fann *ann) {
  struct fann_layer *layer_it;


  printf("Input layer                          :%4d neurons, 1 bias\n", ann->num_input);
  for (layer_it = ann->first_layer + 1; layer_it != ann->last_layer - 1; layer_it++) {
    if (ann->network_type == FANN_NETTYPE_SHORTCUT) {
      printf("  Hidden layer                       :%4d neurons, 0 bias\n",
             (int)(layer_it->last_neuron - layer_it->first_neuron));
    } else {
      printf("  Hidden layer                       :%4d neurons, 1 bias\n",
             (int)(layer_it->last_neuron - layer_it->first_neuron - 1));
    }
  }
  printf("Output layer                         :%4d neurons\n", ann->num_output);
  printf("Total neurons and biases             :%4d\n", fann_get_total_neurons(ann));
  printf("Total connections                    :%4d\n", ann->total_connections);
  printf("Connection rate                      :%8.3f\n", ann->connection_rate);
  printf("Network type                         :   %s\n", FANN_NETTYPE_NAMES[ann->network_type]);

  printf("Training algorithm                   :   %s\n",
         FANN_TRAIN_NAMES[ann->training_algorithm]);
  printf("Training error function              :   %s\n",
         FANN_ERRORFUNC_NAMES[ann->train_error_function]);
  printf("Training stop function               :   %s\n",
         FANN_STOPFUNC_NAMES[ann->train_stop_function]);


  printf("Bit fail limit                       :%8.3f\n", ann->bit_fail_limit);
  printf("Learning rate                        :%8.3f\n", ann->learning_rate);
  printf("Learning momentum                    :%8.3f\n", ann->learning_momentum);
  printf("Quickprop decay                      :%11.6f\n", ann->quickprop_decay);
  printf("Quickprop mu                         :%8.3f\n", ann->quickprop_mu);
  printf("RPROP increase factor                :%8.3f\n", ann->rprop_increase_factor);
  printf("RPROP decrease factor                :%8.3f\n", ann->rprop_decrease_factor);
  printf("RPROP delta min                      :%8.3f\n", ann->rprop_delta_min);
  printf("RPROP delta max                      :%8.3f\n", ann->rprop_delta_max);
  printf("Cascade output change fraction       :%11.6f\n", ann->cascade_output_change_fraction);
  printf("Cascade candidate change fraction    :%11.6f\n", ann->cascade_candidate_change_fraction);
  printf("Cascade output stagnation epochs     :%4d\n", ann->cascade_output_stagnation_epochs);
  printf("Cascade candidate stagnation epochs  :%4d\n", ann->cascade_candidate_stagnation_epochs);
  printf("Cascade max output epochs            :%4d\n", ann->cascade_max_out_epochs);
  printf("Cascade min output epochs            :%4d\n", ann->cascade_min_out_epochs);
  printf("Cascade max candidate epochs         :%4d\n", ann->cascade_max_cand_epochs);
  printf("Cascade min candidate epochs         :%4d\n", ann->cascade_min_cand_epochs);
  printf("Cascade weight multiplier            :%8.3f\n", ann->cascade_weight_multiplier);
  printf("Cascade candidate limit              :%8.3f\n", ann->cascade_candidate_limit);
  
  /* TODO: dump scale parameters */

}


  unsigned int FANN_API fann_get_total_neurons(struct fann *ann) {
  if (ann->network_type) {
    return ann->total_neurons;
  } else {
    /* -1, because there is always an unused bias neuron in the last layer */
    return ann->total_neurons - 1;
  }
}



  enum fann_nettype_enum FANN_API fann_get_network_type(struct fann *ann) {
  /* Currently two types: LAYER = 0, SHORTCUT = 1 */
  /* Enum network_types must be set to match the return values  */
  return ann->network_type;
}

  float FANN_API fann_get_connection_rate(struct fann *ann) {
  return ann->connection_rate;
}

  unsigned int FANN_API fann_get_num_layers(struct fann *ann) {
  return (unsigned int)(ann->last_layer - ann->first_layer);
}

  void FANN_API fann_get_layer_array(struct fann *ann, unsigned int *layers) {
  struct fann_layer *layer_it;

  for (layer_it = ann->first_layer; layer_it != ann->last_layer; layer_it++) {
    unsigned int count = (unsigned int)(layer_it->last_neuron - layer_it->first_neuron);
    /* Remove the bias from the count of neurons. */
    switch (fann_get_network_type(ann)) {
      case FANN_NETTYPE_LAYER: {
        --count;
        break;
      }
      case FANN_NETTYPE_SHORTCUT: {
        /* The bias in the first layer is reused for all layers */
        if (layer_it == ann->first_layer) --count;
        break;
      }
      default: {
        /* Unknown network type, assume no bias present  */
        break;
      }
    }
    *layers++ = count;
  }
}

  void FANN_API fann_get_bias_array(struct fann *ann, unsigned int *bias) {
  struct fann_layer *layer_it;

  for (layer_it = ann->first_layer; layer_it != ann->last_layer; ++layer_it, ++bias) {
    switch (fann_get_network_type(ann)) {
      case FANN_NETTYPE_LAYER: {
        /* Report one bias in each layer except the last */
        if (layer_it != ann->last_layer - 1)
          *bias = 1;
        else
          *bias = 0;
        break;
      }
      case FANN_NETTYPE_SHORTCUT: {
        /* The bias in the first layer is reused for all layers */
        if (layer_it == ann->first_layer)
          *bias = 1;
        else
          *bias = 0;
        break;
      }
      default: {
        /* Unknown network type, assume no bias present  */
        *bias = 0;
        break;
      }
    }
  }
}

  void FANN_API fann_get_connection_array(struct fann *ann,
                                                      struct fann_connection *connections) {
  struct fann_neuron *first_neuron;
  struct fann_layer *layer_it;
  struct fann_neuron *neuron_it;
  unsigned int idx;
  unsigned int source_index;
  unsigned int destination_index;

  first_neuron = ann->first_layer->first_neuron;

  source_index = 0;
  destination_index = 0;

  /* The following assumes that the last unused bias has no connections */

  /* for each layer */
  for (layer_it = ann->first_layer; layer_it != ann->last_layer; layer_it++) {
    /* for each neuron */
    for (neuron_it = layer_it->first_neuron; neuron_it != layer_it->last_neuron; neuron_it++) {
      /* for each connection */
      for (idx = neuron_it->first_con; idx < neuron_it->last_con; idx++) {
        /* Assign the source, destination and weight */
        connections->from_neuron = (unsigned int)(ann->connections[source_index] - first_neuron);
        connections->to_neuron = destination_index;
        connections->weight = ann->weights[source_index];

        connections++;
        source_index++;
      }
      destination_index++;
    }
  }
}

  void FANN_API fann_set_weight_array(struct fann *ann,
                                                  struct fann_connection *connections,
                                                  unsigned int num_connections) {
  unsigned int idx;

  for (idx = 0; idx < num_connections; idx++) {
    fann_set_weight(ann, connections[idx].from_neuron, connections[idx].to_neuron,
                    connections[idx].weight);
  }
}

  void FANN_API fann_set_weight(struct fann *ann, unsigned int from_neuron,
                                            unsigned int to_neuron, fann_type weight) {
  struct fann_neuron *first_neuron;
  struct fann_layer *layer_it;
  struct fann_neuron *neuron_it;
  unsigned int idx;
  unsigned int source_index;
  unsigned int destination_index;

  first_neuron = ann->first_layer->first_neuron;

  source_index = 0;
  destination_index = 0;

  /* Find the connection, simple brute force search through the network
     for one or more connections that match to minimize datastructure dependencies.
     Nothing is done if the connection does not already exist in the network. */

  /* for each layer */
  for (layer_it = ann->first_layer; layer_it != ann->last_layer; layer_it++) {
    /* for each neuron */
    for (neuron_it = layer_it->first_neuron; neuron_it != layer_it->last_neuron; neuron_it++) {
      /* for each connection */
      for (idx = neuron_it->first_con; idx < neuron_it->last_con; idx++) {
        /* If the source and destination neurons match, assign the weight */
        if (((int)from_neuron == ann->connections[source_index] - first_neuron) &&
            (to_neuron == destination_index)) {
          ann->weights[source_index] = weight;
        }
        source_index++;
      }
      destination_index++;
    }
  }
}

  void FANN_API fann_get_weights(struct fann *ann, fann_type *weights) {
  memcpy(weights, ann->weights, sizeof(fann_type) * ann->total_connections);
}

  void FANN_API fann_set_weights(struct fann *ann, fann_type *weights) {
  memcpy(ann->weights, weights, sizeof(fann_type) * ann->total_connections);
}


/* INTERNAL FUNCTION
   Allocates the main structure and sets some default values.
 */
struct fann *fann_allocate_structure(unsigned int num_layers) {
  struct fann *ann;

  if (num_layers < 2) {

    return NULL;
  }

  /* allocate and initialize the main network structure */
  ann = (struct fann *)malloc(sizeof(struct fann));
  if (ann == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  ann->errno_f = FANN_E_NO_ERROR;
  ann->error_log = fann_default_error_log;
  ann->errstr = NULL;
  ann->learning_rate = 0.7f;
  ann->learning_momentum = 0.0;
  ann->total_neurons = 0;
  ann->total_connections = 0;
  ann->num_input = 0;
  ann->num_output = 0;
  ann->train_errors = NULL;
  ann->train_slopes = NULL;
  ann->prev_steps = NULL;
  ann->prev_train_slopes = NULL;
  ann->prev_weights_deltas = NULL;
  ann->training_algorithm = FANN_TRAIN_RPROP;
  ann->num_MSE = 0;
  ann->MSE_value = 0;
  ann->num_bit_fail = 0;
  ann->bit_fail_limit = (fann_type)0.35;
  ann->network_type = FANN_NETTYPE_LAYER;
  ann->train_error_function = FANN_ERRORFUNC_TANH;
  ann->train_stop_function = FANN_STOPFUNC_MSE;
  ann->callback = NULL;
  ann->user_data = NULL; /* User is responsible for deallocation */
  ann->weights = NULL;
  ann->connections = NULL;
  ann->output = NULL;


  /* variables used for cascade correlation (reasonable defaults) */
  ann->cascade_output_change_fraction = 0.01f;
  ann->cascade_candidate_change_fraction = 0.01f;
  ann->cascade_output_stagnation_epochs = 12;
  ann->cascade_candidate_stagnation_epochs = 12;
  ann->cascade_num_candidate_groups = 2;
  ann->cascade_weight_multiplier = (fann_type)0.4;
  ann->cascade_candidate_limit = (fann_type)1000.0;
  ann->cascade_max_out_epochs = 150;
  ann->cascade_max_cand_epochs = 150;
  ann->cascade_min_out_epochs = 50;
  ann->cascade_min_cand_epochs = 50;
  ann->cascade_candidate_scores = NULL;
  ann->cascade_activation_functions_count = 10;
  ann->cascade_activation_functions = (enum fann_activationfunc_enum *)calloc(
      ann->cascade_activation_functions_count, sizeof(enum fann_activationfunc_enum));
  if (ann->cascade_activation_functions == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    free(ann);
    return NULL;
  }

  ann->cascade_activation_functions[0] = FANN_SIGMOID;
  ann->cascade_activation_functions[1] = FANN_SIGMOID_SYMMETRIC;
  ann->cascade_activation_functions[2] = FANN_GAUSSIAN;
  ann->cascade_activation_functions[3] = FANN_GAUSSIAN_SYMMETRIC;
  ann->cascade_activation_functions[4] = FANN_ELLIOT;
  ann->cascade_activation_functions[5] = FANN_ELLIOT_SYMMETRIC;
  ann->cascade_activation_functions[6] = FANN_SIN_SYMMETRIC;
  ann->cascade_activation_functions[7] = FANN_COS_SYMMETRIC;
  ann->cascade_activation_functions[8] = FANN_SIN;
  ann->cascade_activation_functions[9] = FANN_COS;

  ann->cascade_activation_steepnesses_count = 4;
  ann->cascade_activation_steepnesses =
      (fann_type *)calloc(ann->cascade_activation_steepnesses_count, sizeof(fann_type));
  if (ann->cascade_activation_steepnesses == NULL) {
    fann_safe_free(ann->cascade_activation_functions);
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    free(ann);
    return NULL;
  }

  ann->cascade_activation_steepnesses[0] = (fann_type)0.25;
  ann->cascade_activation_steepnesses[1] = (fann_type)0.5;
  ann->cascade_activation_steepnesses[2] = (fann_type)0.75;
  ann->cascade_activation_steepnesses[3] = (fann_type)1.0;

  /* Variables for use with with Quickprop training (reasonable defaults) */
  ann->quickprop_decay = -0.0001f;
  ann->quickprop_mu = 1.75;

  /* Variables for use with with RPROP training (reasonable defaults) */
  ann->rprop_increase_factor = 1.2f;
  ann->rprop_decrease_factor = 0.5;
  ann->rprop_delta_min = 0.0;
  ann->rprop_delta_max = 50.0;
  ann->rprop_delta_zero = 0.1f;

  /* Variables for use with SARPROP training (reasonable defaults) */
  ann->sarprop_weight_decay_shift = -6.644f;
  ann->sarprop_step_error_threshold_factor = 0.1f;
  ann->sarprop_step_error_shift = 1.385f;
  ann->sarprop_temperature = 0.015f;
  ann->sarprop_epoch = 0;

  fann_init_error_data((struct fann_error *)ann);



  /* allocate room for the layers */
  ann->first_layer = (struct fann_layer *)calloc(num_layers, sizeof(struct fann_layer));
  if (ann->first_layer == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    free(ann);
    return NULL;
  }

  ann->last_layer = ann->first_layer + num_layers;

  return ann;
}

/* INTERNAL FUNCTION
   Allocates room for the scaling parameters.
 */
int fann_allocate_scale(struct fann *ann) {
  /* todo this should only be allocated when needed */

  return 0;
}

/* INTERNAL FUNCTION
   Allocates room for the neurons.
 */
void fann_allocate_neurons(struct fann *ann) {
  struct fann_layer *layer_it;
  struct fann_neuron *neurons;
  unsigned int num_neurons_so_far = 0;
  unsigned int num_neurons = 0;

  /* all the neurons is allocated in one long array (calloc clears mem) */
  neurons = (struct fann_neuron *)calloc(ann->total_neurons, sizeof(struct fann_neuron));
  ann->total_neurons_allocated = ann->total_neurons;

  if (neurons == NULL) {
    fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
    return;
  }

  for (layer_it = ann->first_layer; layer_it != ann->last_layer; layer_it++) {
    num_neurons = (unsigned int)(layer_it->last_neuron - layer_it->first_neuron);
    layer_it->first_neuron = neurons + num_neurons_so_far;
    layer_it->last_neuron = layer_it->first_neuron + num_neurons;
    num_neurons_so_far += num_neurons;
  }

  ann->output = (fann_type *)calloc(num_neurons, sizeof(fann_type));
  if (ann->output == NULL) {
    fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
    return;
  }
}

/* INTERNAL FUNCTION
   Allocate room for the connections.
 */
void fann_allocate_connections(struct fann *ann) {
  ann->weights = (fann_type *)calloc(ann->total_connections, sizeof(fann_type));
  if (ann->weights == NULL) {
    fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
    return;
  }
  ann->total_connections_allocated = ann->total_connections;

  /* TODO make special cases for all places where the connections
   * is used, so that it is not needed for fully connected networks.
   */
  ann->connections =
      (struct fann_neuron **)calloc(ann->total_connections_allocated, sizeof(struct fann_neuron *));
  if (ann->connections == NULL) {
    fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
    return;
  }
}


int FANN_SEED_RAND = 1;


  void FANN_API fann_disable_seed_rand() { FANN_SEED_RAND = 0; }

  void FANN_API fann_enable_seed_rand() { FANN_SEED_RAND = 1; }

/* INTERNAL FUNCTION
   Seed the random function.
 */
void fann_seed_rand() {
#ifndef _WIN32
  FILE *fp = fopen("/dev/urandom", "r");
  unsigned int foo;
  struct timeval t;

  if (!fp) {
    gettimeofday(&t, NULL);
    foo = t.tv_usec;
#ifdef DEBUG
    printf("unable to open /dev/urandom\n");
#endif
  } else {
    if (fread(&foo, sizeof(foo), 1, fp) != 1) {
      gettimeofday(&t, NULL);
      foo = t.tv_usec;
#ifdef DEBUG
      printf("unable to read from /dev/urandom\n");
#endif
    }
    fclose(fp);
  }
  if (FANN_SEED_RAND) {
    srand(foo);
  }
#else
  /* COMPAT_TIME REPLACEMENT */
  if (FANN_SEED_RAND) {
    srand(GetTickCount());
  }
#endif
}

/* INTERNAL FUNCTION
   Helper function to update the MSE value and return a diff which takes symmetric functions into
   account
*/
fann_type fann_update_MSE(struct fann *ann, struct fann_neuron *neuron, fann_type neuron_diff) {
  float neuron_diff2;

  switch (neuron->activation_function) {
    case FANN_LINEAR_PIECE_SYMMETRIC:
    case FANN_THRESHOLD_SYMMETRIC:
    case FANN_SIGMOID_SYMMETRIC:
    case FANN_SIGMOID_SYMMETRIC_STEPWISE:
    case FANN_ELLIOT_SYMMETRIC:
    case FANN_GAUSSIAN_SYMMETRIC:
    case FANN_SIN_SYMMETRIC:
    case FANN_COS_SYMMETRIC:
      neuron_diff /= (fann_type)2.0;
      break;
    case FANN_THRESHOLD:
    case FANN_LINEAR:
    case FANN_SIGMOID:
    case FANN_SIGMOID_STEPWISE:
    case FANN_GAUSSIAN:
    case FANN_GAUSSIAN_STEPWISE:
    case FANN_ELLIOT:
    case FANN_LINEAR_PIECE:
    case FANN_SIN:
    case FANN_COS:
      break;
  }


  neuron_diff2 = (float)(neuron_diff * neuron_diff);


  ann->MSE_value += neuron_diff2;

  /*printf("neuron_diff %f = (%f - %f)[/2], neuron_diff2=%f, sum=%f, MSE_value=%f, num_MSE=%d\n",
   * neuron_diff, *desired_output, neuron_value, neuron_diff2, last_layer_begin->sum,
   * ann->MSE_value, ann->num_MSE); */
  if (fann_abs(neuron_diff) >= ann->bit_fail_limit) {
    ann->num_bit_fail++;
  }

  return neuron_diff;
}

/* Tests the network.
 */
  fann_type *FANN_API fann_test(struct fann *ann, fann_type *input,
                                            fann_type *desired_output) {
  fann_type neuron_value;
  fann_type *output_begin = fann_run(ann, input);
  fann_type *output_it;
  const fann_type *output_end = output_begin + ann->num_output;
  fann_type neuron_diff;
  struct fann_neuron *output_neuron = (ann->last_layer - 1)->first_neuron;

  /* calculate the error */
  for (output_it = output_begin; output_it != output_end; output_it++) {
    neuron_value = *output_it;

    neuron_diff = (*desired_output - neuron_value);

    neuron_diff = fann_update_MSE(ann, output_neuron, neuron_diff);

    desired_output++;
    output_neuron++;

    ann->num_MSE++;
  }

  return output_begin;
}

/* get the mean square error.
 */
  float FANN_API fann_get_MSE(struct fann *ann) {
  if (ann->num_MSE) {
    return ann->MSE_value / (float)ann->num_MSE;
  } else {
    return 0;
  }
}

  unsigned int FANN_API fann_get_bit_fail(struct fann *ann) {
  return ann->num_bit_fail;
}

/* reset the mean square error.
 */
  void FANN_API fann_reset_MSE(struct fann *ann) {
  /*printf("resetMSE %d %f\n", ann->num_MSE, ann->MSE_value);*/
  ann->num_MSE = 0;
  ann->MSE_value = 0;
  ann->num_bit_fail = 0;
}


  void FANN_API fann_set_activation_function_hidden(
    struct fann *ann, enum fann_activationfunc_enum activation_function) {
  struct fann_neuron *last_neuron, *neuron_it;
  struct fann_layer *layer_it;
  struct fann_layer *last_layer = ann->last_layer - 1; /* -1 to not update the output layer */

  for (layer_it = ann->first_layer + 1; layer_it != last_layer; layer_it++) {
    last_neuron = layer_it->last_neuron;
    for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
      neuron_it->activation_function = activation_function;
    }
  }
}

  struct fann_layer *FANN_API fann_get_layer(struct fann *ann, int layer) {
  if (layer <= 0 || layer >= (ann->last_layer - ann->first_layer)) {
    fann_error((struct fann_error *)ann, FANN_E_INDEX_OUT_OF_BOUND, layer);
    return NULL;
  }

  return ann->first_layer + layer;
}

  struct fann_neuron *FANN_API fann_get_neuron_layer(struct fann *ann,
                                                                 struct fann_layer *layer,
                                                                 int neuron) {
  if (neuron >= (layer->last_neuron - layer->first_neuron)) {
    fann_error((struct fann_error *)ann, FANN_E_INDEX_OUT_OF_BOUND, neuron);
    return NULL;
  }

  return layer->first_neuron + neuron;
}

  struct fann_neuron *FANN_API fann_get_neuron(struct fann *ann, unsigned int layer,
                                                           int neuron) {
  struct fann_layer *layer_it = fann_get_layer(ann, layer);
  if (layer_it == NULL) return NULL;
  return fann_get_neuron_layer(ann, layer_it, neuron);
}

  enum fann_activationfunc_enum FANN_API fann_get_activation_function(struct fann *ann,
                                                                                  int layer,
                                                                                  int neuron) {
  struct fann_neuron *neuron_it = fann_get_neuron(ann, layer, neuron);
  if (neuron_it == NULL) {
    return (enum fann_activationfunc_enum) - 1; /* layer or neuron out of bounds */
  } else {
    return neuron_it->activation_function;
  }
}

  void FANN_API fann_set_activation_function(
    struct fann *ann, enum fann_activationfunc_enum activation_function, int layer, int neuron) {
  struct fann_neuron *neuron_it = fann_get_neuron(ann, layer, neuron);
  if (neuron_it == NULL) return;

  neuron_it->activation_function = activation_function;
}

  void FANN_API fann_set_activation_function_layer(
    struct fann *ann, enum fann_activationfunc_enum activation_function, int layer) {
  struct fann_neuron *last_neuron, *neuron_it;
  struct fann_layer *layer_it = fann_get_layer(ann, layer);

  if (layer_it == NULL) return;

  last_neuron = layer_it->last_neuron;
  for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
    neuron_it->activation_function = activation_function;
  }
}

  void FANN_API fann_set_activation_function_output(
    struct fann *ann, enum fann_activationfunc_enum activation_function) {
  struct fann_neuron *last_neuron, *neuron_it;
  struct fann_layer *last_layer = ann->last_layer - 1;

  last_neuron = last_layer->last_neuron;
  for (neuron_it = last_layer->first_neuron; neuron_it != last_neuron; neuron_it++) {
    neuron_it->activation_function = activation_function;
  }
}

  void FANN_API fann_set_activation_steepness_hidden(struct fann *ann,
                                                                 fann_type steepness) {
  struct fann_neuron *last_neuron, *neuron_it;
  struct fann_layer *layer_it;
  struct fann_layer *last_layer = ann->last_layer - 1; /* -1 to not update the output layer */

  for (layer_it = ann->first_layer + 1; layer_it != last_layer; layer_it++) {
    last_neuron = layer_it->last_neuron;
    for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
      neuron_it->activation_steepness = steepness;
    }
  }
}

  fann_type FANN_API fann_get_activation_steepness(struct fann *ann, int layer,
                                                               int neuron) {
  struct fann_neuron *neuron_it = fann_get_neuron(ann, layer, neuron);
  if (neuron_it == NULL) {
    return -1; /* layer or neuron out of bounds */
  } else {
    return neuron_it->activation_steepness;
  }
}

  void FANN_API fann_set_activation_steepness(struct fann *ann, fann_type steepness,
                                                          int layer, int neuron) {
  struct fann_neuron *neuron_it = fann_get_neuron(ann, layer, neuron);
  if (neuron_it == NULL) return;

  neuron_it->activation_steepness = steepness;
}

  void FANN_API fann_set_activation_steepness_layer(struct fann *ann,
                                                                fann_type steepness, int layer) {
  struct fann_neuron *last_neuron, *neuron_it;
  struct fann_layer *layer_it = fann_get_layer(ann, layer);

  if (layer_it == NULL) return;

  last_neuron = layer_it->last_neuron;
  for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
    neuron_it->activation_steepness = steepness;
  }
}

  void FANN_API fann_set_activation_steepness_output(struct fann *ann,
                                                                 fann_type steepness) {
  struct fann_neuron *last_neuron, *neuron_it;
  struct fann_layer *last_layer = ann->last_layer - 1;

  last_neuron = last_layer->last_neuron;
  for (neuron_it = last_layer->first_neuron; neuron_it != last_neuron; neuron_it++) {
    neuron_it->activation_steepness = steepness;
  }
}


/*
 * Reads training data from a file.
 */
  struct fann_train_data *FANN_API
fann_read_train_from_file(const char *configuration_file) {
  struct fann_train_data *data;
  FILE *file = fopen(configuration_file, "r");

  if (!file) {
    fann_error(NULL, FANN_E_CANT_OPEN_CONFIG_R, configuration_file);
    return NULL;
  }

  data = fann_read_train_from_fd(file, configuration_file);
  fclose(file);
  return data;
}

/*
 * Save training data to a file
 */
  int FANN_API fann_save_train(struct fann_train_data *data, const char *filename) {
  return fann_save_train_internal(data, filename, 0, 0);
}

/*
 * Save training data to a file in fixed point algebra. (Good for testing
 * a network in fixed point)
 */
  int FANN_API fann_save_train_to_fixed(struct fann_train_data *data,
                                                    const char *filename,
                                                    unsigned int decimal_point) {
  return fann_save_train_internal(data, filename, 1, decimal_point);
}

/*
 * deallocate the train data structure.
 */
  void FANN_API fann_destroy_train(struct fann_train_data *data) {
  if (data == NULL) return;
  if (data->input != NULL) fann_safe_free(data->input[0]);
  if (data->output != NULL) fann_safe_free(data->output[0]);
  fann_safe_free(data->input);
  fann_safe_free(data->output);
  fann_safe_free(data);
}

/*
 * Test a set of training data and calculate the MSE
 */
  float FANN_API fann_test_data(struct fann *ann, struct fann_train_data *data) {
  unsigned int i;
  if (fann_check_input_output_sizes(ann, data) == -1) return 0;

  fann_reset_MSE(ann);

  for (i = 0; i != data->num_data; i++) {
    fann_test(ann, data->input[i], data->output[i]);
  }

  return fann_get_MSE(ann);
}


/*
 * shuffles training data, randomizing the order
 */
  void FANN_API fann_shuffle_train_data(struct fann_train_data *train_data) {
  unsigned int dat = 0, elem, swap;
  fann_type temp;

  for (; dat < train_data->num_data; dat++) {
    swap = (unsigned int)(rand() % train_data->num_data);
    if (swap != dat) {
      for (elem = 0; elem < train_data->num_input; elem++) {
        temp = train_data->input[dat][elem];
        train_data->input[dat][elem] = train_data->input[swap][elem];
        train_data->input[swap][elem] = temp;
      }
      for (elem = 0; elem < train_data->num_output; elem++) {
        temp = train_data->output[dat][elem];
        train_data->output[dat][elem] = train_data->output[swap][elem];
        train_data->output[swap][elem] = temp;
      }
    }
  }
}

/*
 * INTERNAL FUNCTION calculates min and max of train data
 */
void fann_get_min_max_data(fann_type **data, unsigned int num_data, unsigned int num_elem,
                           fann_type *min, fann_type *max) {
  fann_type temp;
  unsigned int dat, elem;
  *min = *max = data[0][0];

  for (dat = 0; dat < num_data; dat++) {
    for (elem = 0; elem < num_elem; elem++) {
      temp = data[dat][elem];
      if (temp < *min)
        *min = temp;
      else if (temp > *max)
        *max = temp;
    }
  }
}

  fann_type FANN_API fann_get_min_train_input(struct fann_train_data *train_data) {
  fann_type min, max;
  fann_get_min_max_data(train_data->input, train_data->num_data, train_data->num_input, &min, &max);
  return min;
}

  fann_type FANN_API fann_get_max_train_input(struct fann_train_data *train_data) {
  fann_type min, max;
  fann_get_min_max_data(train_data->input, train_data->num_data, train_data->num_input, &min, &max);
  return max;
}

  fann_type FANN_API fann_get_min_train_output(struct fann_train_data *train_data) {
  fann_type min, max;
  fann_get_min_max_data(train_data->output, train_data->num_data, train_data->num_output, &min,
                        &max);
  return min;
}

  fann_type FANN_API fann_get_max_train_output(struct fann_train_data *train_data) {
  fann_type min, max;
  fann_get_min_max_data(train_data->output, train_data->num_data, train_data->num_output, &min,
                        &max);
  return max;
}

/*
 * INTERNAL FUNCTION Scales data to a specific range
 */
void fann_scale_data(fann_type **data, unsigned int num_data, unsigned int num_elem,
                     fann_type new_min, fann_type new_max) {
  fann_type old_min, old_max;
  fann_get_min_max_data(data, num_data, num_elem, &old_min, &old_max);
  fann_scale_data_to_range(data, num_data, num_elem, old_min, old_max, new_min, new_max);
}

/*
 * INTERNAL FUNCTION Scales data to a specific range
 */
  void FANN_API fann_scale_data_to_range(fann_type **data, unsigned int num_data,
                                                     unsigned int num_elem, fann_type old_min,
                                                     fann_type old_max, fann_type new_min,
                                                     fann_type new_max) {
  unsigned int dat, elem;
  fann_type temp, old_span, new_span, factor;

  old_span = old_max - old_min;
  new_span = new_max - new_min;
  factor = new_span / old_span;
  /*printf("max %f, min %f, factor %f\n", old_max, old_min, factor);*/

  for (dat = 0; dat < num_data; dat++) {
    for (elem = 0; elem < num_elem; elem++) {
      temp = (data[dat][elem] - old_min) * factor + new_min;
      if (temp < new_min) {
        data[dat][elem] = new_min;
        /*
         * printf("error %f < %f\n", temp, new_min);
         */
      } else if (temp > new_max) {
        data[dat][elem] = new_max;
        /*
         * printf("error %f > %f\n", temp, new_max);
         */
      } else {
        data[dat][elem] = temp;
      }
    }
  }
}

/*
 * Scales the inputs in the training data to the specified range
 */
  void FANN_API fann_scale_input_train_data(struct fann_train_data *train_data,
                                                        fann_type new_min, fann_type new_max) {
  fann_scale_data(train_data->input, train_data->num_data, train_data->num_input, new_min, new_max);
}

/*
 * Scales the inputs in the training data to the specified range
 */
  void FANN_API fann_scale_output_train_data(struct fann_train_data *train_data,
                                                         fann_type new_min, fann_type new_max) {
  fann_scale_data(train_data->output, train_data->num_data, train_data->num_output, new_min,
                  new_max);
}

/*
 * Scales the inputs in the training data to the specified range
 */
  void FANN_API fann_scale_train_data(struct fann_train_data *train_data,
                                                  fann_type new_min, fann_type new_max) {
  fann_scale_data(train_data->input, train_data->num_data, train_data->num_input, new_min, new_max);
  fann_scale_data(train_data->output, train_data->num_data, train_data->num_output, new_min,
                  new_max);
}

/*
 * merges training data into a single struct.
 */
  struct fann_train_data *FANN_API
fann_merge_train_data(struct fann_train_data *data1, struct fann_train_data *data2) {
  unsigned int i;
  fann_type *data_input, *data_output;
  struct fann_train_data *dest = (struct fann_train_data *)malloc(sizeof(struct fann_train_data));

  if (dest == NULL) {
    fann_error((struct fann_error *)data1, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  if ((data1->num_input != data2->num_input) || (data1->num_output != data2->num_output)) {
    fann_error((struct fann_error *)data1, FANN_E_TRAIN_DATA_MISMATCH);
    return NULL;
  }

  fann_init_error_data((struct fann_error *)dest);
  dest->error_log = data1->error_log;

  dest->num_data = data1->num_data + data2->num_data;
  dest->num_input = data1->num_input;
  dest->num_output = data1->num_output;
  dest->input = (fann_type **)calloc(dest->num_data, sizeof(fann_type *));
  if (dest->input == NULL) {
    fann_error((struct fann_error *)data1, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }

  dest->output = (fann_type **)calloc(dest->num_data, sizeof(fann_type *));
  if (dest->output == NULL) {
    fann_error((struct fann_error *)data1, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }

  data_input = (fann_type *)calloc(dest->num_input * dest->num_data, sizeof(fann_type));
  if (data_input == NULL) {
    fann_error((struct fann_error *)data1, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }
  memcpy(data_input, data1->input[0], dest->num_input * data1->num_data * sizeof(fann_type));
  memcpy(data_input + (dest->num_input * data1->num_data), data2->input[0],
         dest->num_input * data2->num_data * sizeof(fann_type));

  data_output = (fann_type *)calloc(dest->num_output * dest->num_data, sizeof(fann_type));
  if (data_output == NULL) {
    fann_error((struct fann_error *)data1, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }
  memcpy(data_output, data1->output[0], dest->num_output * data1->num_data * sizeof(fann_type));
  memcpy(data_output + (dest->num_output * data1->num_data), data2->output[0],
         dest->num_output * data2->num_data * sizeof(fann_type));

  for (i = 0; i != dest->num_data; i++) {
    dest->input[i] = data_input;
    data_input += dest->num_input;
    dest->output[i] = data_output;
    data_output += dest->num_output;
  }
  return dest;
}

/*
 * return a copy of a fann_train_data struct
 */
  struct fann_train_data *FANN_API
fann_duplicate_train_data(struct fann_train_data *data) {
  unsigned int i;
  fann_type *data_input, *data_output;
  struct fann_train_data *dest = (struct fann_train_data *)malloc(sizeof(struct fann_train_data));

  if (dest == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  fann_init_error_data((struct fann_error *)dest);
  dest->error_log = data->error_log;

  dest->num_data = data->num_data;
  dest->num_input = data->num_input;
  dest->num_output = data->num_output;
  dest->input = (fann_type **)calloc(dest->num_data, sizeof(fann_type *));
  if (dest->input == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }

  dest->output = (fann_type **)calloc(dest->num_data, sizeof(fann_type *));
  if (dest->output == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }

  data_input = (fann_type *)calloc(dest->num_input * dest->num_data, sizeof(fann_type));
  if (data_input == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }
  memcpy(data_input, data->input[0], dest->num_input * dest->num_data * sizeof(fann_type));

  data_output = (fann_type *)calloc(dest->num_output * dest->num_data, sizeof(fann_type));
  if (data_output == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }
  memcpy(data_output, data->output[0], dest->num_output * dest->num_data * sizeof(fann_type));

  for (i = 0; i != dest->num_data; i++) {
    dest->input[i] = data_input;
    data_input += dest->num_input;
    dest->output[i] = data_output;
    data_output += dest->num_output;
  }
  return dest;
}

  struct fann_train_data *FANN_API fann_subset_train_data(struct fann_train_data *data,
                                                                      unsigned int pos,
                                                                      unsigned int length) {
  unsigned int i;
  fann_type *data_input, *data_output;
  struct fann_train_data *dest = (struct fann_train_data *)malloc(sizeof(struct fann_train_data));

  if (dest == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  if (pos > data->num_data || pos + length > data->num_data) {
    fann_error((struct fann_error *)data, FANN_E_TRAIN_DATA_SUBSET, pos, length, data->num_data);
    return NULL;
  }

  fann_init_error_data((struct fann_error *)dest);
  dest->error_log = data->error_log;

  dest->num_data = length;
  dest->num_input = data->num_input;
  dest->num_output = data->num_output;
  dest->input = (fann_type **)calloc(dest->num_data, sizeof(fann_type *));
  if (dest->input == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }

  dest->output = (fann_type **)calloc(dest->num_data, sizeof(fann_type *));
  if (dest->output == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }

  data_input = (fann_type *)calloc(dest->num_input * dest->num_data, sizeof(fann_type));
  if (data_input == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }
  memcpy(data_input, data->input[pos], dest->num_input * dest->num_data * sizeof(fann_type));

  data_output = (fann_type *)calloc(dest->num_output * dest->num_data, sizeof(fann_type));
  if (data_output == NULL) {
    fann_error((struct fann_error *)data, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(dest);
    return NULL;
  }
  memcpy(data_output, data->output[pos], dest->num_output * dest->num_data * sizeof(fann_type));

  for (i = 0; i != dest->num_data; i++) {
    dest->input[i] = data_input;
    data_input += dest->num_input;
    dest->output[i] = data_output;
    data_output += dest->num_output;
  }
  return dest;
}

  unsigned int FANN_API fann_length_train_data(struct fann_train_data *data) {
  return data->num_data;
}

  unsigned int FANN_API fann_num_input_train_data(struct fann_train_data *data) {
  return data->num_input;
}

  unsigned int FANN_API fann_num_output_train_data(struct fann_train_data *data) {
  return data->num_output;
}

/* INTERNAL FUNCTION
   Save the train data structure.
 */
int fann_save_train_internal(struct fann_train_data *data, const char *filename,
                             unsigned int save_as_fixed, unsigned int decimal_point) {
  int retval = 0;
  FILE *file = fopen(filename, "w");

  if (!file) {
    fann_error((struct fann_error *)data, FANN_E_CANT_OPEN_TD_W, filename);
    return -1;
  }
  retval = fann_save_train_internal_fd(data, file, filename, save_as_fixed, decimal_point);
  fclose(file);

  return retval;
}

# define FANNPRINTF "%.20e"
/* INTERNAL FUNCTION
   Save the train data structure.
 */
int fann_save_train_internal_fd(struct fann_train_data *data, FILE *file, const char *filename,
                                unsigned int save_as_fixed, unsigned int decimal_point) {
  unsigned int num_data = data->num_data;
  unsigned int num_input = data->num_input;
  unsigned int num_output = data->num_output;
  unsigned int i, j;
  int retval = 0;


  fprintf(file, "%u %u %u\n", data->num_data, data->num_input, data->num_output);

  for (i = 0; i < num_data; i++) {
    for (j = 0; j < num_input; j++) {

      fprintf(file, FANNPRINTF " ", data->input[i][j]);

    }
    fprintf(file, "\n");

    for (j = 0; j < num_output; j++) {

      fprintf(file, FANNPRINTF " ", data->output[i][j]);

    }
    fprintf(file, "\n");
  }

  return retval;
}

/*
 * Creates an empty set of training data
 */
  struct fann_train_data *FANN_API fann_create_train(unsigned int num_data,
                                                                 unsigned int num_input,
                                                                 unsigned int num_output) {
  fann_type *data_input, *data_output;
  unsigned int i;
  struct fann_train_data *data = (struct fann_train_data *)malloc(sizeof(struct fann_train_data));

  if (data == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    return NULL;
  }

  fann_init_error_data((struct fann_error *)data);

  data->num_data = num_data;
  data->num_input = num_input;
  data->num_output = num_output;
  data->input = (fann_type **)calloc(num_data, sizeof(fann_type *));
  if (data->input == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(data);
    return NULL;
  }

  data->output = (fann_type **)calloc(num_data, sizeof(fann_type *));
  if (data->output == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(data);
    return NULL;
  }

  data_input = (fann_type *)calloc(num_input * num_data, sizeof(fann_type));
  if (data_input == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(data);
    return NULL;
  }

  data_output = (fann_type *)calloc(num_output * num_data, sizeof(fann_type));
  if (data_output == NULL) {
    fann_error(NULL, FANN_E_CANT_ALLOCATE_MEM);
    fann_destroy_train(data);
    return NULL;
  }

  for (i = 0; i != num_data; i++) {
    data->input[i] = data_input;
    data_input += num_input;
    data->output[i] = data_output;
    data_output += num_output;
  }
  return data;
}

  struct fann_train_data *FANN_API
fann_create_train_pointer_array(unsigned int num_data, unsigned int num_input, fann_type **input,
                                unsigned int num_output, fann_type **output) {
  unsigned int i;
  struct fann_train_data *data;
  data = fann_create_train(num_data, num_input, num_output);

  if (data == NULL) return NULL;

  for (i = 0; i < num_data; ++i) {
    memcpy(data->input[i], input[i], num_input * sizeof(fann_type));
    memcpy(data->output[i], output[i], num_output * sizeof(fann_type));
  }

  return data;
}

  struct fann_train_data *FANN_API fann_create_train_array(unsigned int num_data,
                                                                       unsigned int num_input,
                                                                       fann_type *input,
                                                                       unsigned int num_output,
                                                                       fann_type *output) {
  unsigned int i;
  struct fann_train_data *data;
  data = fann_create_train(num_data, num_input, num_output);

  if (data == NULL) return NULL;

  for (i = 0; i < num_data; ++i) {
    memcpy(data->input[i], &input[i * num_input], num_input * sizeof(fann_type));
    memcpy(data->output[i], &output[i * num_output], num_output * sizeof(fann_type));
  }

  return data;
}

/*
 * Creates training data from a callback function.
 */
  struct fann_train_data *FANN_API fann_create_train_from_callback(
    unsigned int num_data, unsigned int num_input, unsigned int num_output,
    void(FANN_API *user_function)(unsigned int, unsigned int, unsigned int, fann_type *,
                                  fann_type *)) {
  unsigned int i;
  struct fann_train_data *data = fann_create_train(num_data, num_input, num_output);
  if (data == NULL) {
    return NULL;
  }

  for (i = 0; i != num_data; i++) {
    (*user_function)(i, num_input, num_output, data->input[i], data->output[i]);
  }

  return data;
}

  fann_type *FANN_API fann_get_train_input(struct fann_train_data *data,
                                                       unsigned int position) {
  if (position >= data->num_data) return NULL;
  return data->input[position];
}

  fann_type *FANN_API fann_get_train_output(struct fann_train_data *data,
                                                        unsigned int position) {
  if (position >= data->num_data) return NULL;
  return data->output[position];
}

#define FANNSCANF "%le"

/*
 * INTERNAL FUNCTION Reads training data from a file descriptor.
 */

struct fann_train_data *fann_read_train_from_fd(FILE *file, const char *filename) {
  unsigned int num_input, num_output, num_data, i, j;
  unsigned int line = 1;
  struct fann_train_data *data;

  if (fscanf(file, "%u %u %u\n", &num_data, &num_input, &num_output) != 3) {
    fann_error(NULL, FANN_E_CANT_READ_TD, filename, line);
    return NULL;
  }
  line++;

  data = fann_create_train(num_data, num_input, num_output);
  if (data == NULL) {
    return NULL;
  }

  for (i = 0; i != num_data; i++) {
    for (j = 0; j != num_input; j++) {
      if (fscanf(file, FANNSCANF " ", &data->input[i][j]) != 1) {
        fann_error(NULL, FANN_E_CANT_READ_TD, filename, line);
        fann_destroy_train(data);
        return NULL;
      }
    }
    line++;

    for (j = 0; j != num_output; j++) {
      if (fscanf(file, FANNSCANF " ", &data->output[i][j]) != 1) {
        fann_error(NULL, FANN_E_CANT_READ_TD, filename, line);
        fann_destroy_train(data);
        return NULL;
      }
    }
    line++;
  }
  return data;
}

/*
 * INTERNAL FUNCTION returns 0 if the desired error is reached and -1 if it is not reached
 */
int fann_desired_error_reached(struct fann *ann, float desired_error) {
  switch (ann->train_stop_function) {
    case FANN_STOPFUNC_MSE:
      if (fann_get_MSE(ann) <= desired_error) return 0;
      break;
    case FANN_STOPFUNC_BIT:
      if (ann->num_bit_fail <= (unsigned int)desired_error) return 0;
      break;
  }
  return -1;
}


int fann_check_input_output_sizes(struct fann *ann, struct fann_train_data *data) {
  if (ann->num_input != data->num_input) {
    fann_error((struct fann_error *)ann, FANN_E_INPUT_NO_MATCH, ann->num_input, data->num_input);
    return -1;
  }

  if (ann->num_output != data->num_output) {
    fann_error((struct fann_error *)ann, FANN_E_OUTPUT_NO_MATCH, ann->num_output, data->num_output);
    return -1;
  }

  return 0;
}

float fann_train_epoch_quickprop(struct fann *ann, struct fann_train_data *data) {
  unsigned int i;

  if (ann->prev_train_slopes == NULL) {
    fann_clear_train_arrays(ann);
  }

  fann_reset_MSE(ann);

  for (i = 0; i < data->num_data; i++) {
    fann_run(ann, data->input[i]);
    fann_compute_MSE(ann, data->output[i]);
    fann_backpropagate_MSE(ann);
    fann_update_slopes_batch(ann, ann->first_layer + 1, ann->last_layer - 1);
  }
  fann_update_weights_quickprop(ann, data->num_data, 0, ann->total_connections);

  return fann_get_MSE(ann);
}

/*
 * Internal train function
 */
float fann_train_epoch_irpropm(struct fann *ann, struct fann_train_data *data) {
  unsigned int i;

  if (ann->prev_train_slopes == NULL) {
    fann_clear_train_arrays(ann);
  }

  fann_reset_MSE(ann);
  
  for (i = 0; i < data->num_data; i++) {
    fann_run(ann, data->input[i]);
    fann_compute_MSE(ann, data->output[i]);
    fann_backpropagate_MSE(ann);
    fann_update_slopes_batch(ann, ann->first_layer + 1, ann->last_layer - 1);
  }

  fann_update_weights_irpropm(ann, 0, ann->total_connections);

  return fann_get_MSE(ann);
}

/*
 * Internal train function
 */
float fann_train_epoch_sarprop(struct fann *ann, struct fann_train_data *data) {
  unsigned int i;

  if (ann->prev_train_slopes == NULL) {
    fann_clear_train_arrays(ann);
  }

  fann_reset_MSE(ann);

  for (i = 0; i < data->num_data; i++) {
    fann_run(ann, data->input[i]);
    fann_compute_MSE(ann, data->output[i]);
    fann_backpropagate_MSE(ann);
    fann_update_slopes_batch(ann, ann->first_layer + 1, ann->last_layer - 1);
  }

  fann_update_weights_sarprop(ann, ann->sarprop_epoch, 0, ann->total_connections);

  ++(ann->sarprop_epoch);

  return fann_get_MSE(ann);
}

/*
 * Internal train function
 */
float fann_train_epoch_batch(struct fann *ann, struct fann_train_data *data) {
  unsigned int i;

  fann_reset_MSE(ann);

  for (i = 0; i < data->num_data; i++) {
    fann_run(ann, data->input[i]);
    fann_compute_MSE(ann, data->output[i]);
    fann_backpropagate_MSE(ann);
    fann_update_slopes_batch(ann, ann->first_layer + 1, ann->last_layer - 1);
  }

  fann_update_weights_batch(ann, data->num_data, 0, ann->total_connections);

  return fann_get_MSE(ann);
}

void FANN_API fann_train(struct fann *ann, fann_type *input,
                                       fann_type *desired_output) {
  fann_run(ann, input);

  fann_compute_MSE(ann, desired_output);

  fann_backpropagate_MSE(ann);
  
  fann_update_weights(ann);
}
/*
 * Internal train function
 */
float fann_train_epoch_incremental(struct fann *ann, struct fann_train_data *data) {
  unsigned int i;

  fann_reset_MSE(ann);

  for (i = 0; i != data->num_data; i++) {
    fann_train(ann, data->input[i], data->output[i]);
  }

  return fann_get_MSE(ann);
}

/*
 * Train for one epoch with the selected training algorithm
 */
float FANN_API fann_train_epoch(struct fann *ann, struct fann_train_data *data) {
  if (fann_check_input_output_sizes(ann, data) == -1) return 0;

  switch (ann->training_algorithm) {
    case FANN_TRAIN_QUICKPROP:
      return fann_train_epoch_quickprop(ann, data);
    case FANN_TRAIN_RPROP:
      return fann_train_epoch_irpropm(ann, data);
    case FANN_TRAIN_SARPROP:
      return fann_train_epoch_sarprop(ann, data);
    case FANN_TRAIN_BATCH:
      return fann_train_epoch_batch(ann, data);
    case FANN_TRAIN_INCREMENTAL:
      return fann_train_epoch_incremental(ann, data);
  }
  return 0;
}

void FANN_API fann_train_on_data(struct fann *ann, struct fann_train_data *data,
                                               unsigned int max_epochs,
                                               unsigned int epochs_between_reports,
                                               float desired_error) {
 
  float error;
  unsigned int i=0;
  int desired_error_reached;



  if (epochs_between_reports && ann->callback == NULL) {
    printf("Max epochs %8d. Desired error: %.10f.\n", max_epochs, desired_error);
  }

  for (i = 1; i <= max_epochs; i++) {
    /*
     * train
     */
    error = fann_train_epoch(ann, data);
    desired_error_reached = fann_desired_error_reached(ann, desired_error);

    /*
     * print current output
     */
    if (epochs_between_reports && (i % epochs_between_reports == 0 || i == max_epochs || i == 1 ||
                                   desired_error_reached == 0)) {
      if (ann->callback == NULL) {
        printf("Epochs     %8d. Current error: %.10f. Bit fail %d.\n", i, error, ann->num_bit_fail);
      } else if (((*ann->callback)(ann, data, max_epochs, epochs_between_reports, desired_error,
                                   i)) == -1) {
        /*
         * you can break the training by returning -1
         */
        break;
      }
    }

    if (desired_error_reached == 0) break;
  }
}

void FANN_API fann_set_train_stop_function(struct fann *ann, enum fann_stopfunc_enum stopfunction)
{
  ann->train_stop_function=stopfunction;
}

void FANN_API fann_set_bit_fail_limit(struct fann *ann, fann_type bit_fail)
{
  ann->bit_fail_limit=bit_fail;
}

void FANN_API fann_set_training_algorithm(struct fann *ann, enum fann_train_enum training_algo)
{
  ann->training_algorithm=training_algo;
}

int FANN_API test_callback(struct fann *ann, struct fann_train_data *train,
	unsigned int max_epochs, unsigned int epochs_between_reports, 
	float desired_error, unsigned int epochs)
{
	printf("Epochs     %8d. MSE: %.5f. Desired-MSE: %.5f\n", epochs, fann_get_MSE(ann), desired_error);
	return 0;
}

int main( int argc, char* argv[])
{
fann_type *calc_out;
	unsigned int num_input;
	unsigned int num_output;
	const unsigned int num_layers = 3;
	const unsigned int num_neurons_hidden = 3;
	const float desired_error = 0.001;
	const unsigned int max_epochs = 30;
	const unsigned int epochs_between_reports = 10;
	struct fann *ann;
	struct fann_train_data *data;
  struct fann_train_data *test_data;

	unsigned int i = 0;
	
  data = fann_read_train_from_file(argv[1]);
  num_input=data->num_input;
  num_output=data->num_output;
	printf("Creating network.\n");
 
	ann = fann_create_standard(num_layers, num_input, num_neurons_hidden, num_output);
 
	fann_set_activation_steepness_hidden(ann, 1);
	fann_set_activation_steepness_output(ann, 1);

	fann_set_activation_function_hidden(ann, FANN_SIGMOID_SYMMETRIC);
	fann_set_activation_function_output(ann, FANN_SIGMOID_SYMMETRIC);

  fann_set_train_stop_function(ann, FANN_STOPFUNC_BIT);
	fann_set_bit_fail_limit(ann, 0.01f);

	fann_set_training_algorithm(ann, FANN_TRAIN_RPROP);
	fann_init_weights(ann, data);
	
	printf("Training network.\n");
  fann_print_parameters(ann);
  fann_print_connections(ann);

	fann_train_on_data(ann,data,max_epochs,epochs_between_reports,desired_error);

	fann_destroy_train(data);

	int ret = 0;

	printf("Creating network.\n");

	if(!ann)
	{
		printf("Error creating ann --- ABORTING.\n");
		return -1;
	}

	printf("Testing network.\n");

	test_data = fann_read_train_from_file((argv[2]));

  for(i = 0; i < fann_length_train_data(test_data); i++)
	{
		fann_reset_MSE(ann);
		calc_out = fann_test(ann, test_data->input[i], test_data->output[i]);

		printf("Test (%f, %f) -> %f, should be %f, difference=%f\n",
			   test_data->input[i][0], test_data->input[i][1], calc_out[0], test_data->output[i][0],
			   (float) fann_abs(calc_out[0] - test_data->output[i][0]));
  }
  printf("MSE value for the test data (%f)\n", fann_get_MSE(ann));
	printf("Cleaning up.\n");
	fann_destroy_train(test_data);
	fann_destroy(ann);
  
	return ret;
}
